#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define N 12

using namespace std;

void MostrarMatriz(int filas, int columnas,float* A) {

    for(int i = 0; i < filas; ++i){
        for(int j = 0; j < columnas; ++j){
            cout << A[j * filas + i] << " ";
        }
        cout << endl;
    }
    cout << endl;
}

int main() {
	
	int tamMalloc=N*N* sizeof(float);
	float h_A[N][N]={{147,  67,  56, 124, 151, 111,  89,  24,  39, 141, 151,  52},
	{  8, 146,  74, 101, 112, 150,   9, 174,  67,   3,  48, 102},
	{ 87, 146,  86,  17, 198,  98,  43, 143, 155,  26, 160,  62},
	{187, 115, 104,  24, 152, 118, 138, 168, 193,  12,  61,  87},
	{ 12,  24, 167, 182, 145,  80,  54, 187, 177, 126,  55, 176},
	{ 65,   9,  60, 142,  18,  35, 130, 102, 177,  98, 161, 100},
	{ 24,  83, 178,  37, 195, 110, 140, 131, 158,  90, 141,  66},
	{174, 172,  81, 102,  27,  48, 138,  99, 110,  27,  20,  44},
	{ 89, 163, 150,  37,  27, 166, 120, 140,  36, 185,  63,  81},
	{177, 125, 179,  36,  79,  90, 195, 161, 119, 165,  29, 120},
	{154,   2, 151, 164, 174, 118,  25,  82, 110, 112, 138,  11},
	{152,  78, 111, 132, 176, 181,  34, 133, 155,  14,  88,   9}};
	float h_B[N][N]={{147,  67,  56, 124, 151, 111,  89,  24,  39, 141, 151,  52},
	{  8, 146,  74, 101, 112, 150,   9, 174,  67,   3,  48, 102},
	{ 87, 146,  86,  17, 198,  98,  43, 143, 155,  26, 160,  62},
	{187, 115, 104,  24, 152, 118, 138, 168, 193,  12,  61,  87},
	{ 12,  24, 167, 182, 145,  80,  54, 187, 177, 126,  55, 176},
	{ 65,   9,  60, 142,  18,  35, 130, 102, 177,  98, 161, 100},
	{ 24,  83, 178,  37, 195, 110, 140, 131, 158,  90, 141,  66},
	{174, 172,  81, 102,  27,  48, 138,  99, 110,  27,  20,  44},
	{ 89, 163, 150,  37,  27, 166, 120, 140,  36, 185,  63,  81},
	{177, 125, 179,  36,  79,  90, 195, 161, 119, 165,  29, 120},
	{154,   2, 151, 164, 174, 118,  25,  82, 110, 112, 138,  11},
	{152,  78, 111, 132, 176, 181,  34, 133, 155,  14,  88,   9}};
	float *h_C = (float *)malloc(tamMalloc);

	float *d_A, *d_B, *d_C;
	hipMalloc(&d_A,tamMalloc);
	hipMalloc(&d_B,tamMalloc);
	hipMalloc(&d_C,tamMalloc);

	hipMemcpy(d_A,h_A,tamMalloc,hipMemcpyHostToDevice);
	hipMemcpy(d_B,h_B,tamMalloc,hipMemcpyHostToDevice);

	std::cout << "A =" << std::endl;
	MostrarMatriz
(N, N,(float*) h_A);
	std::cout << "B =" << std::endl;
	MostrarMatriz
(N, N,(float*)h_B);

	float alfa = 1;
	float beta = 0;

	hipblasHandle_t handle;
	hipblasCreate(&handle);

	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alfa, d_A, N, d_B, N, &beta, d_C, N);

	hipblasDestroy(handle);

	hipMemcpy(h_C,d_C,tamMalloc,hipMemcpyDeviceToHost);
	std::cout << "C =" << std::endl;
	MostrarMatriz
( N, N,h_C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	free(h_A);
	free(h_B);
	free(h_C);

	return 0;
}