#include <stdio.h>
#include <unistd.h>
#include "hip/hip_runtime.h"
#include "nccl.h"

#define CUDACHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",             \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define NCCLCHECK(cmd) do {                         \
  ncclResult_t r = cmd;                             \
  if (r!= ncclSuccess) {                            \
    printf("Failed, NCCL error %s:%d '%s'\n",             \
        __FILE__,__LINE__,ncclGetErrorString(r));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)

ncclComm_t* createNccl(int nDevices)
{
	ncclComm_t *comms= new ncclComm_t[nDevices];
	int *devs = new int[nDevices];

	for(int i =0;i<nDevices;i++)
	{
		devs[i]=i;
	}
	NCCLCHECK(ncclCommInitAll(comms, nDevices, devs));
	return comms;
}


int main(int argc, char* argv[])
{
	printf("Empieza \n");
	int nDevicesGlobal;
	CUDACHECK(hipGetDeviceCount(&nDevicesGlobal));
	for(int kk=0;kk<30;kk++)
	{
		sleep(2);

		//managing 4 devices
		int nDev = nDevicesGlobal;
		int size = 32*1024*1024;
		
		ncclComm_t *comms=createNccl(nDevicesGlobal);

		//allocating and initializing device buffers
		float** sendbuff = (float**)malloc(nDev * sizeof(float*));
		float** recvbuff = (float**)malloc(nDev * sizeof(float*));
		hipStream_t* s = (hipStream_t*)malloc(sizeof(hipStream_t)*nDev);


		for (int i = 0; i < nDev; ++i) {
			CUDACHECK(hipSetDevice(i));
			CUDACHECK(hipMalloc(sendbuff + i, size * sizeof(float)));
			CUDACHECK(hipMalloc(recvbuff + i, size * sizeof(float)));
			CUDACHECK(hipMemset(sendbuff[i], 1, size * sizeof(float)));
			CUDACHECK(hipMemset(recvbuff[i], 0, size * sizeof(float)));
			CUDACHECK(hipStreamCreate(s+i));
		}


		//initializing NCCL


		//calling NCCL communication API. Group API is required when using
		//multiple devices per thread
		NCCLCHECK(ncclGroupStart());
		for (int i = 0; i < nDev; ++i)
			NCCLCHECK(ncclAllReduce((const void*)sendbuff[i], (void*)recvbuff[i], size, ncclFloat, ncclSum,
			comms[i], s[i]));
		NCCLCHECK(ncclGroupEnd());


		//synchronizing on CUDA streams to wait for completion of NCCL operation
		for (int i = 0; i < nDev; ++i) {
		CUDACHECK(hipSetDevice(i));
		CUDACHECK(hipStreamSynchronize(s[i]));
		}


		//free device buffers
		for (int i = 0; i < nDev; ++i) {
		CUDACHECK(hipSetDevice(i));
		CUDACHECK(hipFree(sendbuff[i]));
		CUDACHECK(hipFree(recvbuff[i]));
		CUDACHECK(hipStreamDestroy(s[i]));

		}


		//finalizing NCCL
		for(int i = 0; i < nDev; ++i)
		{
			ncclCommDestroy(comms[i]);
		}


		printf("Success; %d\n",kk);
	}
	sleep(2);
	return 0;
}