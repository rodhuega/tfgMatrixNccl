#include "NcclMultiplicationEnvironment.cuh"


template <class Toperation>
NcclMultiplicationEnvironment<Toperation>::NcclMultiplicationEnvironment(int gpuSizeWorld,int gpuRoot,OperationType opType)
{
    this->gpuRoot=gpuRoot;

    CUDACHECK(hipGetDeviceCount(&gpuSizeSystem));

    if(gpuSizeWorld!=-1)
    {
        this->gpuSizeWorld=gpuSizeWorld;
    }else
    {
        this->gpuSizeWorld=gpuSizeSystem;
    }
    this->opType=opType;
    if(opType==MultDouble)
    {
        basicOperationType=ncclDouble;
    }else
    {
        basicOperationType=ncclFloat;
    }
    std::cout<<"System gpus: "<<this->gpuSizeSystem<<". World gpus: "<<this->gpuSizeWorld<<std::endl;
}


template class NcclMultiplicationEnvironment<double>;
