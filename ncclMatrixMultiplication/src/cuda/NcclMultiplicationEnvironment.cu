#include "hip/hip_runtime.h"
#include "NcclMultiplicationEnvironment.cuh"


template <class Toperation>
NcclMultiplicationEnvironment<Toperation>::NcclMultiplicationEnvironment(int gpuSizeWorld,int gpuRoot,OperationType opType,bool printMatrix)
{
    this->gpuSizeOperationWorld=-1;
    this->gpuRoot=gpuRoot;
    this->printMatrix=printMatrix;
    this->lastMeshRowSize=-1;
    this->lastMeshColumnSize=-1;
    this->lastBlockRowSizeA=-1;
    this->lastBlockColumnSizeA=-1;
    this->lastBlockRowSizeB=-1;
    this->lastBlockColumnSizeB=-1;
    CUDACHECK(hipGetDeviceCount(&gpuSizeSystem));

    if(gpuSizeWorld!=-1)
    {
        this->gpuSizeWorld=max(gpuSizeWorld,4);
    }else
    {
        this->gpuSizeWorld=max(gpuSizeSystem,4);
    }
    this->opType=opType;
    if(opType==MultDouble)
    {
        basicOperationType=ncclDouble;
    }else
    {
        basicOperationType=ncclFloat;
    }

    //Crear un cublasHandler con su stream correspondiente por cada gpu física del sistema
    int i;
    for(i=0;i<gpuSizeSystem;i++)
    {
        CUDACHECK(hipSetDevice(i));
        hipStream_t *newStream = new hipStream_t;
        hipblasHandle_t *newHandle = new hipblasHandle_t;
        CUDACHECK(hipStreamCreate(newStream));
        cublasStreams.push_back(newStream);
        CUBLASCHECK(hipblasCreate(newHandle));
        CUBLASCHECK(hipblasSetStream(*newHandle,*newStream));
        cublasHandlers.push_back(newHandle);
    }
    
    //SOLO DEBUG
    if(printMatrix)
    {
        std::cout<<"System gpus: "<<this->gpuSizeSystem<<". World gpus: "<<this->gpuSizeWorld<<std::endl;
    }

}

template <class Toperation>
NcclMultiplicationEnvironment<Toperation>::~NcclMultiplicationEnvironment()
{
    int i;
    
    //Eliminar los elementos del map que contiene los comunicadores
    for(auto itMap=summaComms.begin();itMap!=summaComms.end();itMap++)
    {
        auto actualElement=itMap->second;
        std::vector<CommSummaElement*> commActual=std::get<0>(actualElement);
        for(i=0;i<commActual.size();i++)
        {
            delete commActual[i];
        }
        commActual.clear();
        std::get<1>(actualElement).clear();
        std::get<2>(actualElement).clear();
    }
    summaComms.clear();
    eraseBufferMatrix();
    for (i=0;i<cublasStreams.size();i++)
    {
        CUDACHECK(hipSetDevice(i));
        CUBLASCHECK(hipblasDestroy(*cublasHandlers[i]));
        CUDACHECK(hipStreamDestroy(*cublasStreams[i]));
        delete cublasHandlers[i];
        delete cublasStreams[i];
        // CUDACHECK(hipDeviceReset());
    }
    cublasStreams.clear();
    cublasHandlers.clear();
    
}

template <class Toperation>
ncclDataType_t NcclMultiplicationEnvironment<Toperation>::getBasicOperationType()
{
    return basicOperationType;
}

template <class Toperation>
void NcclMultiplicationEnvironment<Toperation>::eraseBufferMatrix()
{
    int i;
    for(i=0;i<gpuAuxiliarMatricesA.size() || i<gpuAuxiliarMatricesB.size();i++)
    {
        CUDACHECK(hipSetDevice(MatrixUtilitiesCuda<Toperation>::getRealGpuId(i,gpuSizeSystem)));
        if(i<gpuAuxiliarMatricesA.size())
        {
            MatrixUtilitiesCuda<Toperation>::matrixFreeGPU(gpuAuxiliarMatricesA[i]);
        }
        if(i<gpuAuxiliarMatricesB.size())
        {
            MatrixUtilitiesCuda<Toperation>::matrixFreeGPU(gpuAuxiliarMatricesB[i]);
        }
    }
    gpuAuxiliarMatricesA.clear();
    gpuAuxiliarMatricesB.clear();
}

template <class Toperation>
int NcclMultiplicationEnvironment<Toperation>::getGpuSizeOperationWorld()
{
    return gpuSizeOperationWorld;
}

template <class Toperation>
int NcclMultiplicationEnvironment<Toperation>::getGpuSizeOperationSystem()
{
    return gpuSizeOperationSystem;
}

template <class Toperation>
int NcclMultiplicationEnvironment<Toperation>::getGpuSizeSystem()
{
    return gpuSizeSystem;
}

template <class Toperation>
int NcclMultiplicationEnvironment<Toperation>::getGpuSizeWorld()
{
    return gpuSizeWorld;
}

template <class Toperation>
int NcclMultiplicationEnvironment<Toperation>::getGpuRoot()
{
    return gpuRoot;
}

template <class Toperation>
OperationType NcclMultiplicationEnvironment<Toperation>::getOperationType()
{
    return opType;
}

template <class Toperation>
std::vector<hipblasHandle_t*> NcclMultiplicationEnvironment<Toperation>::getCublasHandlers()
{
    return cublasHandlers;
}

template <class Toperation>
void NcclMultiplicationEnvironment<Toperation>::setGpuSizeOperationWorld(int gpuSizeOperationWorld)
{
    this->gpuSizeOperationWorld=gpuSizeOperationWorld;
}
template <class Toperation>
void NcclMultiplicationEnvironment<Toperation>::setGpuSizeOperationSystem(int gpuSizeOperationSystem)
{
    this->gpuSizeOperationSystem=gpuSizeOperationSystem;
}

template <class Toperation>
void NcclMultiplicationEnvironment<Toperation>::waitAllCublasStreams()
{
    int i;
    for(i=0;i<cublasStreams.size();i++)
    {
        CUDACHECK(hipSetDevice(i));
        CUDACHECK(hipStreamSynchronize(*cublasStreams[i]));
    }
}

template <class Toperation>
void NcclMultiplicationEnvironment<Toperation>::createNcclCommunicator(std::vector<CommSummaElement*> &commElements,std::set<int> &dimensionLogicDevices,bool setRowColor)
{
    int i,j,k,rank,gpuIdPhysical,logicRankIndex=0;
    //Vector que contiene los rangos de las gpus que acompañaran a esa gpu en el comunicador
    std::vector<int> logicRanks(gpuSizeOperationWorld);

    //Vector que contiene las gpus físicas que formaran parte del comunicador para cada grupo. Primer vector son las gpus físicas
    std::vector<std::vector<int>> devicesOfComm;
    std::vector<std::vector<int>> logicDevices;
    //Vector que en la posicion i tiene todas las gpus lógicas asociadas a esa física para cada grupo. Primer vector son las gpus físicas
    std::vector<std::vector<std::vector<int>>> physicalToLogic;
    bool assigned=false;
    for(int gpuIdLogic: dimensionLogicDevices)
    {
        gpuIdPhysical=commElements[gpuIdLogic]->getIdPhysical();
        for(i=0;i<devicesOfComm.size();i++)
        {   //Ver si se puede agregar como gpu física
            if (std::find(devicesOfComm[0].begin(), devicesOfComm[0].end(),gpuIdPhysical ) == devicesOfComm[0].end()) {
                devicesOfComm[0].push_back(gpuIdPhysical);
                assigned=true;
                physicalToLogic[0][gpuIdPhysical].push_back(gpuIdLogic);
                logicDevices[0].push_back(gpuIdLogic);
            }
        }

        if(!assigned)
        {//En caso de que no no haya gpu física para esa gpu lógica
            std::vector<int> newDevicesOfComm;newDevicesOfComm.push_back(gpuIdPhysical);
            std::vector<std::vector<int>> newPhysicalToLogic(gpuSizeOperationSystem);newPhysicalToLogic[gpuIdPhysical].push_back(gpuIdLogic);
            std::vector<int> newLogicDevices;newLogicDevices.push_back(gpuIdLogic);
            logicDevices.push_back(newLogicDevices);
            physicalToLogic.push_back(newPhysicalToLogic);
            devicesOfComm.push_back(newDevicesOfComm);
        }
        assigned=false;
        logicRanks[gpuIdLogic]=logicRankIndex;
        logicRankIndex++;
    }
    //Creación del comunicador y asignacion al elemento correspondiente del vector commElements
    for(j=0;j<devicesOfComm.size();j++)
    {
        ncclComm_t* newComm= new ncclComm_t[devicesOfComm[j].size()];    
        NCCLCHECK(ncclCommInitAll(newComm, devicesOfComm[j].size(), &devicesOfComm[j][0]));
        for(i=0;i<devicesOfComm[j].size();i++)
        {
            hipSetDevice(devicesOfComm[j][i]);
            ncclCommUserRank(newComm[i],&rank);
            
            std::vector<int> auxDevicesOfComm=devicesOfComm[j];
            std::vector<std::vector<int>> auxPhysicalToLogic=physicalToLogic[j];
            for(int gpuIdLogic:auxPhysicalToLogic[auxDevicesOfComm[i]])
            {
                if(setRowColor)
                {
                    commElements[gpuIdLogic]->setRankCommRowLogic(logicRanks[gpuIdLogic]);
                    commElements[gpuIdLogic]->setRankCommRowPhysical(rank);
                    commElements[gpuIdLogic]->setCommRow(newComm[i]);
                    if(logicDevices[0][0]!=logicDevices[j][0])
                    {
                        for(k=0;k<logicDevices.size();k++)
                        {//En caso de que no no haya gpu física para esa gpu lógica
                            for(int ll=0;ll<logicDevices[k].size();ll++)
                            {
                                commElements[logicDevices[k][ll]]->addCommRowMySelf(newComm[i]);
                                commElements[gpuIdLogic]->addCommRowMySelf(newComm[i]);
                            }
                        }
                    }
                    commElements[gpuIdLogic]->setRowDevices(logicDevices);
                }else
                {
                    commElements[gpuIdLogic]->setRankCommColumnLogic(logicRanks[gpuIdLogic]);
                    commElements[gpuIdLogic]->setRankCommColumnPhysical(rank);
                    commElements[gpuIdLogic]->setCommColumn(newComm[i]);
                    if(logicDevices[0][0]!=logicDevices[j][0])
                    {
                        for(k=0;k<logicDevices.size();k++)
                        {//En caso de que no no haya gpu física para esa gpu lógica
                            for(int ll=0;ll<logicDevices[k].size();ll++)
                            {
                                commElements[logicDevices[k][ll]]->addCommColumnMySelf(newComm[i]);
                                commElements[gpuIdLogic]->addCommColumnMySelf(newComm[i]);
                            }
                        }
                    }
                    commElements[gpuIdLogic]->setColumnDevices(logicDevices);
                }
            }
        }
    }
}

template <class Toperation>
std::tuple<std::vector<CommSummaElement*>,std::vector<std::set<int>>,std::vector<std::set<int>>> NcclMultiplicationEnvironment<Toperation>::getOrCreateCommunicators(int meshRowSize, int meshColumnSize,MatrixMain<Toperation>* matrixA)
{
    auto itMapComm= summaComms.find(meshRowSize);
    if(itMapComm==summaComms.end())
    {
        //Sets en el que cada elemeneto es el color y tienen un vector de las ids lógica de los elementos que pertenecen a ese color. Usado dentro del bucle de Summa porque find O(log n)
        std::vector<std::set<int>> rowColorsLogic(meshRowSize),columnColorsLogic(meshColumnSize);
        //Array de vecotores que tendra los comunicadores(array de ncclComm_t) de cada gpu lógica
        std::vector<CommSummaElement*> commElements(gpuSizeOperationWorld);
        int i,rowColor,columnColor;
        //Inicialización de los vectores descritos previamente para tener la información necesaria para realizar Summa
        for(i=0;i<gpuSizeOperationWorld;i++)
        {
            int gpuRealId=MatrixUtilitiesCuda<Toperation>::getRealGpuId(i,gpuSizeSystem);
            rowColor=matrixA->calculateRowColor(i);
            columnColor=matrixA->calculateColumnColor(i);
            rowColorsLogic[rowColor].insert(i);
            columnColorsLogic[columnColor].insert(i);
            commElements[i]=new CommSummaElement(i,gpuRealId,rowColor,columnColor);
        }
        //Creación de los comunicadores
        std::set<int> rowsColorSet,columnColorSet;
        for(i=0;i<meshRowSize||i<meshColumnSize;i++)
        {
            if(i<meshRowSize)
            {
                rowsColorSet = rowColorsLogic[i];
                createNcclCommunicator(commElements,rowsColorSet,true);
            }
            if(i<meshColumnSize)
            {
                columnColorSet = columnColorsLogic[i];
                createNcclCommunicator(commElements,columnColorSet,false);
            }
        }
        summaComms[meshRowSize]=std::make_tuple(commElements,rowColorsLogic,columnColorsLogic);
    }
    return summaComms[meshRowSize];
}


template <class Toperation>
MatrixMain<Toperation>& NcclMultiplicationEnvironment<Toperation>::performCalculations(MatrixMain<Toperation>& ma,MatrixMain<Toperation>& mb)
{
    OperationProperties op;
    MatrixMain<Toperation> *mc;

    if(!MatrixUtilitiesCuda<Toperation>::canMultiply(ma.getColumnsReal(),mb.getRowsReal()))
    {
        throw std::invalid_argument("La operación no se puede realizar porque las columnas no coinciden con las filas. Columnas: " +std::to_string(ma.getColumnsReal())+ ", Filas: "+ std::to_string(mb.getRowsReal()));
    }

    //METER AQUÍ COMPROBACIÖN DE TAMAÑO Y SI ES MENOR HACERLA SECUENCIAL EN CASO DE QUE SE QUIERA HACER

    //Realización de la distribución pertinente
    if(ma.getIsDistributed()&&mb.getIsDistributed()&&ma.getBlockColumnSize()==mb.getBlockRowSize())
    {
        op.meshRowSize=ma.getMeshRowSize();
        op.meshColumnSize=ma.getMeshColumnSize();
    }else if(ma.getIsDistributed() && !mb.getIsDistributed())
    {
        op=MatrixUtilitiesCuda<Toperation>::getMeshAndMatrixSizeFromOneDistributedMatrix(ma.getRowsUsed(),ma.getColumnsUsed(), mb.getRowsReal(),mb.getColumnsReal(),ma.getMeshRowSize(),ma.getMeshColumnSize(),true);
        mb.setRowsUsed(op.columnsAorRowsB);
        mb.setColumnsUsed(op.columnsB);
        mb.setMatrixOperationProperties(op.meshRowSize,op.meshColumnSize,op.blockRowSizeB,op.blockColumnSizeB);
        mb.distributeMatrixIntoGpus();
        mb.waitAllStreamsOfAllWorkers();
    }else if(!ma.getIsDistributed() && mb.getIsDistributed())
    {
        op=MatrixUtilitiesCuda<Toperation>::getMeshAndMatrixSizeFromOneDistributedMatrix(ma.getRowsReal(),ma.getColumnsReal(), mb.getRowsUsed(),mb.getColumnsUsed(),mb.getMeshRowSize(),mb.getMeshColumnSize(),false);
        ma.setRowsUsed(op.rowsA);
        ma.setColumnsUsed(op.columnsAorRowsB);
        ma.setMatrixOperationProperties(op.meshRowSize,op.meshColumnSize,op.blockRowSizeA,op.blockColumnSizeA);
        ma.distributeMatrixIntoGpus();
        ma.waitAllStreamsOfAllWorkers();
    }else if(ma.getIsDistributed()&& mb.getIsDistributed()&&ma.getBlockColumnSize()!=mb.getBlockRowSize())
    {//Se decide recuperar b y redistribuirla. Puede que haya una mejor estrategia
        mb.getHostMatrix();
        mb.setIsDistributed(false);
        op=MatrixUtilitiesCuda<Toperation>::getMeshAndMatrixSizeFromOneDistributedMatrix(ma.getRowsUsed(),ma.getColumnsUsed(), mb.getRowsReal(),mb.getColumnsReal(),ma.getMeshRowSize(), ma.getMeshColumnSize(),true);
        mb.setRowsUsed(op.columnsAorRowsB);
        mb.setColumnsUsed(op.columnsB);
        mb.setMatrixOperationProperties(op.meshRowSize,op.meshColumnSize,op.blockRowSizeB,op.blockColumnSizeB);
        mb.distributeMatrixIntoGpus();
        mb.waitAllStreamsOfAllWorkers();
    }else if(!ma.getIsDistributed() && !mb.getIsDistributed())
    {
        op = MatrixUtilitiesCuda<Toperation>::getMeshAndMatrixSize(ma.getRowsReal(), ma.getColumnsReal(), mb.getRowsReal(), mb.getColumnsReal(), gpuSizeWorld);
        
        ma.setRowsUsed(op.rowsA);
        ma.setColumnsUsed(op.columnsAorRowsB);
        mb.setRowsUsed(op.columnsAorRowsB);
        mb.setColumnsUsed(op.columnsB);

        if (printMatrix)
        {
            std::cout << "NGpus: " << op.gpuSize << ", meshRowSize: " << op.meshRowSize << ", meshColumnSize: " << op.meshColumnSize << ", blockRowSizeA: " << \
            op.blockRowSizeA << ", blockColumnSizeA: " << op.blockColumnSizeA << ", blockRowSizeB: " << op.blockRowSizeB << ", blockColumnSizeB: " << \
            op.blockColumnSizeB << ", rowsA: " << op.rowsA << ", columnsAorRowsB: " << op.columnsAorRowsB << ", columnsB: " << op.columnsB << std::endl;

            std::cout << "A-> Rows: " << ma.getRowsReal() << ", Columns: " << ma.getColumnsReal() << ", Matriz A:" << std::endl;
            MatrixUtilitiesCuda<Toperation>::printMatrix(ma.getRowsReal(), ma.getColumnsReal(), ma.getHostMatrix());
            std::cout << "B-> Rows: " << mb.getRowsReal() << ", Columns: " << mb.getColumnsReal() << ", Matriz B:" << std::endl;
            MatrixUtilitiesCuda<Toperation>::printMatrix(mb.getRowsReal(), mb.getColumnsReal(), mb.getHostMatrix());
        }
        this->gpuSizeOperationWorld=op.gpuSize;
        this->gpuSizeOperationSystem=min(this->gpuSizeSystem,op.gpuSize);
        ma.setMatrixOperationProperties(op.meshRowSize,op.meshColumnSize,op.blockRowSizeA,op.blockColumnSizeA);
        mb.setMatrixOperationProperties(op.meshRowSize,op.meshColumnSize,op.blockRowSizeB,op.blockColumnSizeB);
        ma.distributeMatrixIntoGpus();
        mb.distributeMatrixIntoGpus();
        ma.waitAllStreamsOfAllWorkers();
        mb.waitAllStreamsOfAllWorkers();
    }else
    {
        throw std::runtime_error("Error. Se ha producido algún error en la librería");
    }

    mc=ncclSumma(&ma,&mb,op.meshRowSize,op.meshColumnSize);
    return *mc;
}

template <class Toperation>
MatrixMain<Toperation>*  NcclMultiplicationEnvironment<Toperation>::ncclSumma(MatrixMain<Toperation>* matrixA, MatrixMain<Toperation>* matrixB, int meshRowsSize, int meshColumnsSize)
{
    int i,vecI,gpuRank,rootRank;
    std::vector<int> vecOfActualComm;
    ncclComm_t commActual;hipStream_t *streamComm;
    int rowsA = matrixA->getRowsUsed();
    int columnsAorRowsB = matrixA->getColumnsUsed();
    int columnsB = matrixB->getColumnsUsed();
    int blockSizeA = matrixA->getBlockSize();
    int blockSizeB = matrixB->getBlockSize();
    int blockRowSizeA = matrixA->getBlockRowSize();
    int blockColumnsSizeA = matrixA->getBlockColumnSize();
    int blockColumnsSizeB = matrixB->getBlockColumnSize();
    int blockRowSizeB = matrixB->getBlockRowSize();
    Toperation alphaGemm= matrixA->getAlphaGemm();
    //Creación del esquelo del elemento que va a ser devuelto
    MatrixMain<Toperation> *mc= new MatrixMain<Toperation>(this,matrixA->getRowsReal(),matrixB->getColumnsReal());
    mc->setRowsUsed(matrixA->getRowsUsed());
    mc->setColumnsUsed(matrixB->getColumnsUsed());
    mc->setMatrixOperationProperties(meshRowsSize,meshColumnsSize,blockRowSizeA,blockColumnsSizeB);
    mc->setIsDistributed(true);
    //Crear o recuperar los comunicadores
    auto commData=getOrCreateCommunicators(meshRowsSize,meshColumnsSize,matrixA);
    std::vector<CommSummaElement*> commElements=std::get<0>(commData);
    std::vector<std::set<int>> rowColorsLogic=std::get<1>(commData);
    std::vector<std::set<int>> columnColorsLogic=std::get<2>(commData);

    if(lastMeshRowSize!=meshRowsSize && lastMeshColumnSize!= meshColumnsSize && lastBlockRowSizeA!=blockRowSizeA 
        && lastBlockColumnSizeA!=blockColumnsSizeA  && lastBlockRowSizeB!=blockRowSizeB && lastBlockColumnSizeB!=blockColumnsSizeB )
    {
        eraseBufferMatrix();
        //Reserva de las matrices buffer para cada gpu
        for(i=0;i<gpuSizeOperationWorld;i++)
        {
            int gpuRealId=MatrixUtilitiesCuda<Toperation>::getRealGpuId(i,gpuSizeSystem);
            CUDACHECK(hipSetDevice(gpuRealId));
            Toperation *gpuAuxA=MatrixUtilitiesCuda<Toperation>::cudaMatrixMemoryAllocationGPU(blockRowSizeA,blockColumnsSizeA,cublasStreams[gpuRealId]);
            Toperation *gpuAuxB=MatrixUtilitiesCuda<Toperation>::cudaMatrixMemoryAllocationGPU(blockRowSizeB,blockColumnsSizeB,cublasStreams[gpuRealId]);
            gpuAuxiliarMatricesA.push_back(gpuAuxA);gpuAuxiliarMatricesB.push_back(gpuAuxB);
        }
        lastMeshRowSize=meshRowsSize; lastMeshColumnSize=meshColumnsSize;
        lastBlockRowSizeA=blockRowSizeA; lastBlockColumnSizeA=blockColumnsSizeA;
        lastBlockRowSizeB=blockRowSizeB; lastBlockColumnSizeB=blockColumnsSizeB;
    }
    
    //Realizacion de las operaciones matematicas. Algoritmo Summa
    for (i = 0; i < meshColumnsSize; i++)
    {
        //Copiar las matrices que tocan al buffer
        for(gpuRank=0;gpuRank<gpuSizeOperationWorld;gpuRank++)
        {
            int gpuRealId=MatrixUtilitiesCuda<Toperation>::getRealGpuId(gpuRank,gpuSizeSystem);
            CUDACHECK(hipSetDevice(gpuRealId));
            if (columnColorsLogic[(i % meshColumnsSize)].find(gpuRank)!=columnColorsLogic[(i % meshColumnsSize)].end())
            {
                CUDACHECK(hipMemcpyAsync(gpuAuxiliarMatricesA[gpuRank],matrixA->getGpuWorkers()[gpuRank]->getMatrixLocal(i / meshColumnsSize),blockSizeA*sizeof(Toperation),hipMemcpyDeviceToDevice,*matrixA->getGpuWorkers()[gpuRank]->getStream(i / meshColumnsSize)));
            }
            if (rowColorsLogic[(i % meshRowsSize)].find(gpuRank)!=rowColorsLogic[(i % meshRowsSize)].end())
            {
                CUDACHECK(hipMemcpyAsync(gpuAuxiliarMatricesB[gpuRank],matrixB->getGpuWorkers()[gpuRank]->getMatrixLocal(i / meshRowsSize),blockSizeB*sizeof(Toperation),hipMemcpyDeviceToDevice,*matrixB->getGpuWorkers()[gpuRank]->getStream(i / meshRowsSize)));
            }
        }
        //Esperar esa copia
        matrixA->waitAllStreamsOfAllWorkers();
        matrixB->waitAllStreamsOfAllWorkers();
        
        //Realizacion de las comunicaciones
        NCCLCHECK(ncclGroupStart());
        for(gpuRank=0;gpuRank<gpuSizeOperationWorld;gpuRank++)
	    {
            if(commElements[gpuRank]->getRankCommRowLogic()==(i % meshColumnsSize))
            {
                for(vecI=0;vecI<commElements[gpuRank]->getRowDevices().size();vecI++)
                {
                    vecOfActualComm=commElements[gpuRank]->getRowDevices()[vecI];
                    if(std::find(vecOfActualComm.begin(), vecOfActualComm.end(),gpuRank ) == vecOfActualComm.end())
                    {//Para gpus lógicas que no están físicas
                        vecOfActualComm.push_back(gpuRank);
                    }
                    for(int gpuIdComm:vecOfActualComm)
                    {
                        int realId=MatrixUtilitiesCuda<Toperation>::getRealGpuId(gpuIdComm,gpuSizeSystem);
                        CUDACHECK(hipSetDevice(realId));
                        streamComm=commElements[gpuIdComm]->getStreamRow();
                        commActual=commElements[gpuIdComm]->getCommRow();
                        rootRank=commElements[gpuRank]->getRankCommRowPhysical();
                        if(MatrixUtilitiesCuda<Toperation>::getRealGpuId(vecOfActualComm[0],gpuSizeSystem)!=vecOfActualComm[0]&& vecI>0)
                        {//Para gpus lógicas que no están físicas
                            streamComm=commElements[gpuIdComm]->getStreamRowMySelf();
                            commActual=commElements[gpuIdComm]->getCommRowMySelf();
                            rootRank=0;
                        }
                        NCCLCHECK(ncclBroadcast(gpuAuxiliarMatricesA[gpuRank],gpuAuxiliarMatricesA[gpuIdComm],blockSizeA,
                            basicOperationType,rootRank,commActual,
                            *streamComm));
                    }
                }
            }
            if(commElements[gpuRank]->getRankCommColumnLogic()==(i % meshRowsSize))
            {
                for(vecI=0;vecI<commElements[gpuRank]->getColumnDevices().size();vecI++)
                {
                    vecOfActualComm=commElements[gpuRank]->getColumnDevices()[vecI];
                    if(std::find(vecOfActualComm.begin(), vecOfActualComm.end(),gpuRank ) == vecOfActualComm.end())
                    {//Para gpus lógicas que no están físicas
                        vecOfActualComm.push_back(gpuRank);
                    }
                    for(int gpuIdComm:vecOfActualComm)
                    {
                        int realId=MatrixUtilitiesCuda<Toperation>::getRealGpuId(gpuIdComm,gpuSizeSystem);
                        CUDACHECK(hipSetDevice(realId));
                        streamComm=commElements[gpuIdComm]->getStreamColumn();
                        commActual=commElements[gpuIdComm]->getCommColumn();
                        rootRank=commElements[gpuRank]->getRankCommColumnPhysical();
                        if(MatrixUtilitiesCuda<Toperation>::getRealGpuId(vecOfActualComm[0],gpuSizeSystem)!=vecOfActualComm[0]&& vecI>0)
                        {//Para gpus lógicas que no están físicas. Importante el orden de estas dos instrucciones para que vaya bien el índice
                            streamComm=commElements[gpuIdComm]->getStreamColumnMySelf();
                            commActual=commElements[gpuIdComm]->getCommColumnMySelf();
                            rootRank=0;
                        }
                        NCCLCHECK(ncclBroadcast(gpuAuxiliarMatricesB[gpuRank],gpuAuxiliarMatricesB[gpuIdComm],blockSizeB,
                            basicOperationType,rootRank,commActual,
                            *streamComm));
                    }
                }
            }
        }
        NCCLCHECK(ncclGroupEnd());
        //Esperar las comunicaciones
        for(gpuRank=0;gpuRank<commElements.size();gpuRank++)
        {
            CUDACHECK(hipSetDevice(MatrixUtilitiesCuda<Toperation>::getRealGpuId(gpuRank,gpuSizeSystem)));
            commElements[gpuRank]->waitStreams();
        }
        //Realización de todas las multiplicaciones
        for(gpuRank=0;gpuRank<gpuSizeOperationWorld;gpuRank++)
	    {
            int gpuRealId=MatrixUtilitiesCuda<Toperation>::getRealGpuId(gpuRank,gpuSizeSystem);
            CUDACHECK(hipSetDevice(gpuRealId));
            MatrixUtilitiesCuda<Toperation>::matrixCublasMultiplication(cublasHandlers[gpuRealId],opType,blockRowSizeA,blockRowSizeB,blockColumnsSizeB,gpuAuxiliarMatricesA[gpuRank],gpuAuxiliarMatricesB[gpuRank],mc->getGpuWorkers()[gpuRank]->getMatrixLocal(0),alphaGemm,1.0);
        }
        waitAllCublasStreams();
    }
    return mc;
}

template class NcclMultiplicationEnvironment<double>;
template class NcclMultiplicationEnvironment<float>;

