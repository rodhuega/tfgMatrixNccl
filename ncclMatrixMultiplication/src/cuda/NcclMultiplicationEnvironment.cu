#include "hip/hip_runtime.h"
#include "NcclMultiplicationEnvironment.cuh"


template <class Toperation>
NcclMultiplicationEnvironment<Toperation>::NcclMultiplicationEnvironment(int gpuSizeWorld,int gpuRoot,OperationType opType)
{
    this->gpuSizeOperationWorld=-1;
    this->gpuRoot=gpuRoot;

    CUDACHECK(hipGetDeviceCount(&gpuSizeSystem));

    if(gpuSizeWorld!=-1)
    {
        this->gpuSizeWorld=gpuSizeWorld;
    }else
    {
        this->gpuSizeWorld=gpuSizeSystem;
    }
    this->opType=opType;
    if(opType==MultDouble)
    {
        basicOperationType=ncclDouble;
    }else
    {
        basicOperationType=ncclFloat;
    }
    std::cout<<"System gpus: "<<this->gpuSizeSystem<<". World gpus: "<<this->gpuSizeWorld<<std::endl;

    //Crear un cublasHandler con su stream correspondiente por cada gpu física del sistema
    int i;
    for(i=0;i<gpuSizeSystem;i++)
    {
        CUDACHECK(hipSetDevice(i));
        hipStream_t *newStream = new hipStream_t;
        hipblasHandle_t *newHandle = new hipblasHandle_t;
        CUDACHECK(hipStreamCreate(newStream));
        cublasStreams.push_back(newStream);
        CUBLASCHECK(hipblasCreate(newHandle));
        CUBLASCHECK(hipblasSetStream(*newHandle,*newStream));
        cublasHandlers.push_back(newHandle);
    }
}

template <class Toperation>
int NcclMultiplicationEnvironment<Toperation>::getGpuSizeOperationWorld()
{
    return gpuSizeOperationWorld;
}
template <class Toperation>
int NcclMultiplicationEnvironment<Toperation>::getGpuSizeOperationSystem()
{
    return gpuSizeOperationSystem;
}
template <class Toperation>
int NcclMultiplicationEnvironment<Toperation>::getGpuSizeSystem()
{
    return gpuSizeSystem;
}
template <class Toperation>
int NcclMultiplicationEnvironment<Toperation>::getGpuSizeWorld()
{
    return gpuSizeWorld;
}
template <class Toperation>
int NcclMultiplicationEnvironment<Toperation>::getGpuRoot()
{
    return gpuRoot;
}
template <class Toperation>
void NcclMultiplicationEnvironment<Toperation>::waitAllCublasStreams()
{
    int i;
    for(i=0;i<cublasStreams.size();i++)
    {
        CUDACHECK(hipSetDevice(i));
        CUDACHECK(hipStreamSynchronize(*cublasStreams[i]));
    }
}

template <class Toperation>
void NcclMultiplicationEnvironment<Toperation>::setOrAddMatrixMain(std::string id, MatrixMain<Toperation> *matrixMainGlobal)
{
    matricesMatrixMain[id] = matrixMainGlobal;
}

template <class Toperation>//////////////////Por revisar esta implementacion
MatrixMain<Toperation> *NcclMultiplicationEnvironment<Toperation>::getMainMatrix(std::string id, bool create)
{
    auto it = matricesMatrixMain.find(id);
    if (it == matricesMatrixMain.end())
    {
        if (!create)
        {
            throw std::invalid_argument("La matriz global existe");
        }
        else
        {
            // Toperation *matrixAux = getMatrixGlobalSimplePointer(id);
            // dimensions dimensionsAux = matricesGlobalDimensions[id];
            // MatrixMain<Toperation> *res = new MatrixMain<Toperation>(std::get<0>(dimensionsAux), std::get<1>(dimensionsAux));
            // setOrAddMatrixMain(id, res);
            // return res;
        }
    }
    return it->second;
}

template <class Toperation>
void NcclMultiplicationEnvironment<Toperation>::removeMatrixMain(std::string id,bool freeMemory)
{
    MatrixMain<Toperation> * auxMatrix=matricesMatrixMain[id];
    matricesMatrixMain.erase(id);
    if(freeMemory)
    {
        delete auxMatrix;
    }
}

template <class Toperation>
void NcclMultiplicationEnvironment<Toperation>::createNcclCommunicator(std::vector<CommSummaElement*> &commElements,std::set<int> &dimensionLogicDevices,bool setRowColor)
{
    int i,rank,gpuIdPhysical,logicRankIndex=0;
    std::vector<int> logicRanks(gpuSizeOperationWorld);
    std::vector<int> devicesOfComm;
    std::vector<int> logicDevices;
    // Array que en la posicion i tiene todas las gpus Logicas asociadas a esa Fisica
    std::vector<std::vector<int>> physicalToLogic(gpuSizeOperationSystem);
    for(int gpuIdLogic: dimensionLogicDevices)
    {
        gpuIdPhysical=commElements[gpuIdLogic]->getIdPhysical();
        if (std::find(devicesOfComm.begin(), devicesOfComm.end(),gpuIdPhysical ) == devicesOfComm.end()) {
            devicesOfComm.push_back(gpuIdPhysical);
        }
        physicalToLogic[gpuIdPhysical].push_back(gpuIdLogic);
        logicDevices.push_back(gpuIdLogic);
        logicRanks[gpuIdLogic]=logicRankIndex;
        logicRankIndex++;
    }
    ncclComm_t* newComm= new ncclComm_t[devicesOfComm.size()];
    // NCCLCHECK(ncclCommInitAll(newComm, devicesOfComm.size(), &devicesOfComm[0]));
    
    NCCLCHECK(ncclCommInitAll(newComm, devicesOfComm.size(), &devicesOfComm[0]));
    for(i=0;i<devicesOfComm.size();i++)
    {
        hipSetDevice(devicesOfComm[i]);
        ncclCommUserRank(newComm[i],&rank);
        //Setear 
        for(int gpuIdLogic:physicalToLogic[devicesOfComm[i]])
        {
            if(setRowColor)
            {
                commElements[gpuIdLogic]->setRankCommRowPhysical(rank);
                commElements[gpuIdLogic]->setRankCommRowLogic(logicRanks[gpuIdLogic]);
                commElements[gpuIdLogic]->setCommRow(newComm[i]);
                commElements[gpuIdLogic]->setRowDevices(logicDevices);
            }else
            {
                commElements[gpuIdLogic]->setRankCommColumnPhysical(rank);
                commElements[gpuIdLogic]->setRankCommColumnLogic(logicRanks[gpuIdLogic]);
                commElements[gpuIdLogic]->setCommColumn(newComm[i]);
                commElements[gpuIdLogic]->setColumnDevices(logicDevices);
            }
        }
    }
}

template <class Toperation>
void NcclMultiplicationEnvironment<Toperation>::setCommOperation(int gpuOperationSize)
{
    //Caso de que no se puede reutilizar el comunicador
    if(this->gpuSizeOperationWorld!=gpuOperationSize)
    {
        int i;
        if(this->gpuSizeOperationWorld!=-1)
        {
            for (i = 0; i < gpuSizeOperationSystem; ++i)
	        {
                NCCLCHECK(ncclCommDestroy(commOperation[i]));
	        }
        }
        
        this->gpuSizeOperationWorld=gpuOperationSize;
        
        gpuSizeOperationSystem=min(gpuSizeOperationWorld,gpuSizeSystem);
        int arrayGpuSystemCommOperation[gpuSizeOperationSystem];
        for(i=0;i<gpuSizeOperationSystem;i++)
        {
            arrayGpuSystemCommOperation[i]=i;
        }
        commOperation=new ncclComm_t[gpuSizeOperationSystem];
        NCCLCHECK(ncclCommInitAll(commOperation, gpuSizeOperationSystem, arrayGpuSystemCommOperation));
    }
}

template <class Toperation>
std::string NcclMultiplicationEnvironment<Toperation>::generateRandomCandiateId()
{
    std::string str("0123456789ABCDEFGHIJKLMNOPQRSTUVWXYZabcdefghijklmnopqrstuvwxyz");

    std::random_device rd;
    std::mt19937 generator(rd());

    std::shuffle(str.begin(), str.end(), generator);

    return str.substr(0, 8);
}

template <class Toperation>
std::string NcclMultiplicationEnvironment<Toperation>::generateRandomId()
{
    bool randomGenerated=false;
    std::string id;
    while(!randomGenerated)
    {
        id=generateRandomCandiateId();
        auto it = matricesMatrixMain.find(id);
        if (it == matricesMatrixMain.end())
        {
            randomGenerated=true;
        }
    }
    return id;
}

template <class Toperation>
std::vector<int> NcclMultiplicationEnvironment<Toperation>::convertSetToVector(std::set<int> &s)
{
    std::vector<int> v(s.size());
    std::copy(s.begin(), s.end(), v.begin());
    return v;
}

template <class Toperation>
MatrixMain<Toperation> *NcclMultiplicationEnvironment<Toperation>::performCalculations(std::string idA,std::string idB, std::string idC,bool printMatrix)
{
    OperationProperties op;
    MatrixMain<Toperation> *ma, *mb, *mc;
    ma=getMainMatrix(idA,false);
    mb=getMainMatrix(idB,false);

    if(!MatrixUtilities<Toperation>::canMultiply(ma->getColumnsReal(),mb->getRowsReal()))
    {
        throw std::invalid_argument("La operacion no se puede realizar porque las columnas no coinciden con las filas. Columnas: " +std::to_string(ma->getColumnsReal())+ ", Filas: "+ std::to_string(mb->getRowsReal()));
    }

    if(!ma->getIsDistributed() && !mb->getIsDistributed())
    {
        op = MatrixUtilities<double>::getMeshAndMatrixSize(ma->getRowsReal(), ma->getColumnsReal(), mb->getRowsReal(), mb->getColumnsReal(), gpuSizeWorld);
        std::cout << "NGpus: " << op.gpuSize << ", meshRowSize: " << op.meshRowSize << ", meshColumnSize: " << op.meshColumnSize << ", blockRowSizeA: " << \
            op.blockRowSizeA << ", blockColumnSizeA: " << op.blockColumnSizeA << ", blockRowSizeB: " << op.blockRowSizeB << ", blockColumnSizeB: " << \
            op.blockColumnSizeB << ", rowsA: " << op.rowsA << ", columnsAorRowsB: " << op.columnsAorRowsB << ", columnsB: " << op.columnsB << std::endl;
        
        ma->setRowsUsed(op.rowsA);
        ma->setColumnsUsed(op.columnsAorRowsB);
        
        mb->setRowsUsed(op.columnsAorRowsB);
        mb->setColumnsUsed(op.columnsB);

        if (printMatrix)
        {
            std::cout << "A-> Rows: " << ma->getRowsReal() << ", Columns: " << ma->getColumnsReal() << ", Matriz A:" << std::endl;
            MatrixUtilities<Toperation>::printMatrix(ma->getRowsReal(), ma->getColumnsReal(), ma->getHostMatrix());
            std::cout << "B-> Rows: " << mb->getRowsReal() << ", Columns: " << mb->getColumnsReal() << ", Matriz B:" << std::endl;
            MatrixUtilities<Toperation>::printMatrix(mb->getRowsReal(), mb->getColumnsReal(), mb->getHostMatrix());
        }
        
        setCommOperation(op.gpuSize);
        ma->setMatrixOperationProperties(op.meshRowSize,op.meshColumnSize,op.blockRowSizeA,op.blockColumnSizeA);
        mb->setMatrixOperationProperties(op.meshRowSize,op.meshColumnSize,op.blockRowSizeB,op.blockColumnSizeB);
        ma->distributeMatrixIntoGpus();
        mb->distributeMatrixIntoGpus();
        ma->waitAllStreamsOfAllWorkers();
        mb->waitAllStreamsOfAllWorkers();

        // MatrixUtilitiesCuda<Toperation>::cudaDebugMatricesLocalDifferentGpuWorkers(gpuSizeOperationWorld,ma->getBlockRowSize(),ma->getBlockColumnSize(),ma->getGpuWorkers());

        mc=mpiSumma(ma,mb,op.meshRowSize,op.meshColumnSize);
        if(idC!="")
        {
            mc->setId(idC);
        }
        // MatrixUtilitiesCuda<Toperation>::cudaDebugMatricesLocalDifferentGpuWorkers(gpuSizeOperationWorld,mc->getBlockRowSize(),mc->getBlockColumnSize(),mc->getGpuWorkers());
    }
    return mc;
}

template <class Toperation>
MatrixMain<Toperation>*  NcclMultiplicationEnvironment<Toperation>::mpiSumma(MatrixMain<Toperation>* matrixA, MatrixMain<Toperation>* matrixB, int meshRowsSize, int meshColumnsSize)
{
    int i,j,gpuRank,gpuRealIdCommRowRoot,gpuRealIdCommColumnRoot,gpuLogicIdCommRowRoot,gpuLogicIdCommColumnRoot,rowColor,columnColor;
    int rowsA = matrixA->getRowsUsed();
    int columnsAorRowsB = matrixA->getColumnsUsed();
    int columnsB = matrixB->getColumnsUsed();
    int blockSizeA = matrixA->getBlockSize();
    int blockSizeB = matrixB->getBlockSize();
    int blockRowSizeA = matrixA->getBlockRowSize();
    int blockColumnsSizeA = matrixA->getBlockColumnSize();
    int blockColumnsSizeB = matrixB->getBlockColumnSize();
    int blockRowSizeB = matrixB->getBlockRowSize();
    //Creacion del esquelo del elemento que va a ser devuelto
    MatrixMain<Toperation> *mc= new MatrixMain<Toperation>(this,generateRandomId(),matrixA->getRowsReal(),matrixB->getColumnsReal());
    mc->setIsDistributed(true);
    mc->setRowsUsed(matrixA->getRowsUsed());
    mc->setColumnsUsed(matrixB->getColumnsUsed());
    mc->setMatrixOperationProperties(meshRowsSize,meshColumnsSize,blockRowSizeA,blockColumnsSizeB);
    //Reserva de las matrices buffer para cada gpu y conseguir a que columna y fila pertenece cada gpu. Posicion i de los vectores asociadas a esa i de gpuWorker
    std::vector<Toperation*> gpuAuxiliarMatricesA,gpuAuxiliarMatricesB;
    //Sets que para cada elemento que indica el color tienen un vector de la id logica de los elementos que pertenecen a ese color
    std::vector<std::set<int>> rowColorsLogic(meshRowsSize),columnColorsLogic(meshColumnsSize);
    //Sets para crear los comunicadores con la gpu fisica
    std::vector<std::set<int>> rowColorPhysicalSet(meshRowsSize),columnColorPhysicalSet(meshColumnsSize);
    //Array de vecotores que tendra los comunicadores(array de ncclComm_t) de cada gpu logica
    std::vector<CommSummaElement*> commElements(gpuSizeOperationWorld);
    for(i=0;i<gpuSizeOperationWorld;i++)
    {
        int gpuRealId=MatrixUtilitiesCuda<Toperation>::getRealGpuId(i,gpuSizeSystem);
        CUDACHECK(hipSetDevice(gpuRealId));
        Toperation *gpuAuxA=MatrixUtilitiesCuda<Toperation>::cudaMatrixMemoryAllocation(blockRowSizeA,blockColumnsSizeA,cublasStreams[gpuRealId]);
        Toperation *gpuAuxB=MatrixUtilitiesCuda<Toperation>::cudaMatrixMemoryAllocation(blockRowSizeB,blockColumnsSizeB,cublasStreams[gpuRealId]);
        gpuAuxiliarMatricesA.push_back(gpuAuxA);gpuAuxiliarMatricesB.push_back(gpuAuxB);
        rowColor=matrixA->calculateRowColor(i);
        columnColor=matrixA->calculateColumnColor(i);
        rowColorsLogic[rowColor].insert(i);
        columnColorsLogic[columnColor].insert(i);

        commElements[i]=new CommSummaElement(i,gpuRealId,rowColor,columnColor);
        rowColorPhysicalSet[rowColor].insert(gpuRealId);
        columnColorPhysicalSet[columnColor].insert(gpuRealId);
    }
    //Creacion de los comunicadores
    std::set<int> rowsColorSet,columnColorSet;
    for(i=0;i<meshRowsSize||i<meshColumnsSize;i++)
    {
        rowsColorSet = rowColorsLogic[i];
        columnColorSet = columnColorsLogic[i];
        if(i<meshRowsSize)
        {
            createNcclCommunicator(commElements,rowsColorSet,true);

        }
        if(i<meshColumnsSize)
        {
            createNcclCommunicator(commElements,columnColorSet,false);
        }
    }

    //Realizacion de las operaciones matematicas
    for (i = 0; i < meshColumnsSize; i++)
    {
        //Copiar las matrices que tocan al buffer
        for(gpuRank=0;gpuRank<gpuSizeOperationWorld;gpuRank++)
        {
            int gpuRealId=MatrixUtilitiesCuda<Toperation>::getRealGpuId(gpuRank,gpuSizeSystem);
            CUDACHECK(hipSetDevice(gpuRealId));
            if (columnColorsLogic[(i % meshColumnsSize)].find(gpuRank)!=columnColorsLogic[(i % meshColumnsSize)].end())
            {
                // memcpy(matrixAuxiliarA, matrixLocalA.getMatrixLocal(i / meshColumnsSize), blockSizeA * sizeof(Toperation));
                CUDACHECK(hipMemcpyAsync(gpuAuxiliarMatricesA[gpuRank],matrixA->getGpuWorkers()[gpuRank]->getMatrixLocal(i / meshColumnsSize),blockSizeA*sizeof(Toperation),hipMemcpyDeviceToDevice,*matrixA->getGpuWorkers()[gpuRank]->getStream(i / meshColumnsSize)));
            }
            if (rowColorsLogic[(i % meshRowsSize)].find(gpuRank)!=rowColorsLogic[(i % meshRowsSize)].end())
            {
                CUDACHECK(hipMemcpyAsync(gpuAuxiliarMatricesB[gpuRank],matrixB->getGpuWorkers()[gpuRank]->getMatrixLocal(i / meshColumnsSize),blockSizeB*sizeof(Toperation),hipMemcpyDeviceToDevice,*matrixB->getGpuWorkers()[gpuRank]->getStream(i / meshColumnsSize)));
                // memcpy(matrixAuxiliarB, matrixLocalB.getMatrixLocal(i / meshRowsSize), blockSizeB * sizeof(Toperation));
            }
        }
        gpuRealIdCommRowRoot=MatrixUtilitiesCuda<Toperation>::getRealGpuId((i % meshColumnsSize),gpuSizeSystem);
        gpuRealIdCommColumnRoot=MatrixUtilitiesCuda<Toperation>::getRealGpuId((i % meshRowsSize),gpuSizeSystem);
        gpuLogicIdCommRowRoot=i % meshColumnsSize;
        gpuLogicIdCommColumnRoot=i % meshRowsSize;
        //Esperar esa copia
        matrixA->waitAllStreamsOfAllWorkers();
        matrixB->waitAllStreamsOfAllWorkers();
        
        //Realizacion de las comunicaciones
        std::vector<std::vector<hipStream_t*>> commStreams(gpuSizeOperationWorld);
        NCCLCHECK(ncclGroupStart());
        for(gpuRank=0;gpuRank<gpuSizeOperationWorld;gpuRank++)
	    {
            if(commElements[gpuRank]->getRankCommRowLogic()==(i % meshRowsSize))
            {
                for(int gpuRankComm:commElements[gpuRank]->getRowDevices())
                {
                    CUDACHECK(hipSetDevice(MatrixUtilitiesCuda<Toperation>::getRealGpuId(gpuRankComm,gpuSizeSystem)));
                    hipStream_t* newStream = new hipStream_t;
                    commStreams[gpuRankComm].push_back(newStream);
                    CUDACHECK(hipStreamCreate(newStream));
                    NCCLCHECK(ncclBroadcast(gpuAuxiliarMatricesA[gpuRank],gpuAuxiliarMatricesA[gpuRankComm],blockSizeA,
                        basicOperationType,commElements[gpuRank]->getRankCommRowPhysical(),commElements[gpuRankComm]->getCommRow(),
                        *newStream));
                }
            }
            if(commElements[gpuRank]->getRankCommColumnLogic()==(i % meshColumnsSize))
            {
                for(int gpuRankComm:commElements[gpuRank]->getColumnDevices())
                {
                    CUDACHECK(hipSetDevice(MatrixUtilitiesCuda<Toperation>::getRealGpuId(gpuRankComm,gpuSizeSystem)));
                    hipStream_t* newStream = new hipStream_t;
                    commStreams[gpuRankComm].push_back(newStream);

                    CUDACHECK(hipStreamCreate(newStream));
                    NCCLCHECK(ncclBroadcast(gpuAuxiliarMatricesB[gpuRank],gpuAuxiliarMatricesB[gpuRankComm],blockSizeB,
                        basicOperationType,commElements[gpuRank]->getRankCommColumnPhysical(),commElements[gpuRankComm]->getCommColumn(),
                        *newStream));
                }
            }
        }
        NCCLCHECK(ncclGroupEnd());
        //Esperar las comunicaciones
        for(gpuRank=0;gpuRank<gpuSizeOperationWorld;gpuRank++)
        {
            CUDACHECK(hipSetDevice(MatrixUtilitiesCuda<Toperation>::getRealGpuId(gpuRank,gpuSizeSystem)));
            for(j=0;j<commStreams[gpuRank].size();j++)
            {
                CUDACHECK(hipStreamSynchronize(*commStreams[gpuRank][j]));
            }
        }
        
        for(gpuRank=0;gpuRank<gpuSizeOperationWorld;gpuRank++)
	    {
            // std::cout<<"Aux A: Iteracion: "<<i<<", gpuRank: "<<gpuRank<<std::endl;
            // MatrixUtilitiesCuda<Toperation>::cudaPrintOneMatrixCall(matrixA->getBlockRowSize(),matrixA->getBlockColumnSize(),gpuAuxiliarMatricesA[gpuRank]);
            // std::cout<<"Aux B: Iteracion: "<<i<<", gpuRank: "<<gpuRank<<std::endl;
            // MatrixUtilitiesCuda<Toperation>::cudaPrintOneMatrixCall(matrixB->getBlockRowSize(),matrixB->getBlockColumnSize(),gpuAuxiliarMatricesB[gpuRank]);
            int gpuRealId=MatrixUtilitiesCuda<Toperation>::getRealGpuId(gpuRank,gpuSizeSystem);
            CUDACHECK(hipSetDevice(gpuRealId));
            MatrixUtilitiesCuda<Toperation>::matrixCublasMultiplication(cublasHandlers[gpuRealId],blockRowSizeA,blockRowSizeB,blockColumnsSizeB,gpuAuxiliarMatricesA[gpuRank],gpuAuxiliarMatricesB[gpuRank],mc->getGpuWorkers()[gpuRank]->getMatrixLocal(0));
        }
        waitAllCublasStreams();
        // std::cout<<"Iteracion: "<<i<<std::endl;
        // MatrixUtilitiesCuda<Toperation>::cudaDebugMatricesLocalDifferentGpuWorkers(gpuSizeOperationWorld,mc->getBlockRowSize(),mc->getBlockColumnSize(),mc->getGpuWorkers());
    }
    // //Liberacion de las matrices auxiliares que realizaban computo
    // MatrixUtilities<Toperation>::matrixFree(matrixAuxiliarA);
    // MatrixUtilities<Toperation>::matrixFree(matrixAuxiliarB);
    // return matrixLocalC;

    //FALTA LIBERAR MEMORIA
    return mc;
    
}


template class NcclMultiplicationEnvironment<double>;
