#include "hip/hip_runtime.h"
#include "NcclMultiplicationEnvironment.cuh"


template <class Toperation>
NcclMultiplicationEnvironment<Toperation>::NcclMultiplicationEnvironment(int gpuSizeWorld,int gpuRoot,OperationType opType)
{
    this->gpuSizeOperationWorld=-1;
    this->gpuRoot=gpuRoot;

    CUDACHECK(hipGetDeviceCount(&gpuSizeSystem));

    if(gpuSizeWorld!=-1)
    {
        this->gpuSizeWorld=gpuSizeWorld;
    }else
    {
        this->gpuSizeWorld=gpuSizeSystem;
    }
    this->opType=opType;
    if(opType==MultDouble)
    {
        basicOperationType=ncclDouble;
    }else
    {
        basicOperationType=ncclFloat;
    }
    std::cout<<"System gpus: "<<this->gpuSizeSystem<<". World gpus: "<<this->gpuSizeWorld<<std::endl;
}

template <class Toperation>
int NcclMultiplicationEnvironment<Toperation>::getGpuSizeOperationWorld()
{
    return gpuSizeOperationWorld;
}
template <class Toperation>
int NcclMultiplicationEnvironment<Toperation>::getGpuSizeOperationSystem()
{
    return gpuSizeOperationSystem;
}
template <class Toperation>
int NcclMultiplicationEnvironment<Toperation>::getGpuSizeSystem()
{
    return gpuSizeSystem;
}
template <class Toperation>
int NcclMultiplicationEnvironment<Toperation>::getGpuSizeWorld()
{
    return gpuSizeWorld;
}
template <class Toperation>
int NcclMultiplicationEnvironment<Toperation>::getGpuRoot()
{
    return gpuRoot;
}

template <class Toperation>
void NcclMultiplicationEnvironment<Toperation>::setOrAddMatrixMain(std::string id, MatrixMain<Toperation> *matrixMainGlobal)
{
    matricesMatrixMain[id] = matrixMainGlobal;
}

template <class Toperation>//////////////////Por revisar esta implementacion
MatrixMain<Toperation> *NcclMultiplicationEnvironment<Toperation>::getMainMatrix(std::string id, bool create)
{
    auto it = matricesMatrixMain.find(id);
    if (it == matricesMatrixMain.end())
    {
        if (!create)
        {
            throw std::invalid_argument("La matriz global existe");
        }
        else
        {
            // Toperation *matrixAux = getMatrixGlobalSimplePointer(id);
            // dimensions dimensionsAux = matricesGlobalDimensions[id];
            // MatrixMain<Toperation> *res = new MatrixMain<Toperation>(std::get<0>(dimensionsAux), std::get<1>(dimensionsAux));
            // setOrAddMatrixMain(id, res);
            // return res;
        }
    }
    return it->second;
}

template <class Toperation>
void NcclMultiplicationEnvironment<Toperation>::setCommOperation(int gpuOperationSize)
{
    //Caso de que no se puede reutilizar el comunicador
    if(this->gpuSizeOperationWorld!=gpuOperationSize)
    {
        int i;
        if(this->gpuSizeOperationWorld!=-1)
        {
            for (i = 0; i < gpuSizeOperationSystem; ++i)
	        {
                NCCLCHECK(ncclCommDestroy(commOperation[i]));
	        }
        }
        
        this->gpuSizeOperationWorld=gpuOperationSize;
        
        gpuSizeOperationSystem=min(gpuSizeOperationWorld,gpuSizeSystem);
        int arrayGpuSystemCommOperation[gpuSizeOperationSystem];
        for(i=0;i<gpuSizeOperationSystem;i++)
        {
            arrayGpuSystemCommOperation[i]=i;
        }
        commOperation=new ncclComm_t[gpuSizeOperationSystem];
        NCCLCHECK(ncclCommInitAll(commOperation, gpuSizeOperationSystem, arrayGpuSystemCommOperation));
    }
}

template <class Toperation>
void NcclMultiplicationEnvironment<Toperation>::performCalculations(std::string idA,std::string idB, std::string idC,bool printMatrix)
{
    OperationProperties op;
    MatrixMain<Toperation> *ma, *mb, *mc;
    ma=getMainMatrix(idA,false);
    mb=getMainMatrix(idB,false);

    if(!MatrixUtilities<Toperation>::canMultiply(ma->getColumnsReal(),mb->getRowsReal()))
    {
        throw std::invalid_argument("La operacion no se puede realizar porque las columnas no coinciden con las filas. Columnas: " +std::to_string(ma->getColumnsReal())+ ", Filas: "+ std::to_string(mb->getRowsReal()));
    }

    if(!ma->getIsDistributed() && !mb->getIsDistributed())
    {
        op = MatrixUtilities<double>::getMeshAndMatrixSize(ma->getRowsReal(), ma->getColumnsReal(), mb->getRowsReal(), mb->getColumnsReal(), gpuSizeWorld);
        std::cout << "NGpus: " << op.gpuSize << ", meshRowSize: " << op.meshRowSize << ", meshColumnSize: " << op.meshColumnSize << ", blockRowSizeA: " << \
            op.blockRowSizeA << ", blockColumnSizeA: " << op.blockColumnSizeA << ", blockRowSizeB: " << op.blockRowSizeB << ", blockColumnSizeB: " << \
            op.blockColumnSizeB << ", rowsA: " << op.rowsA << ", columnsAorRowsB: " << op.columnsAorRowsB << ", columnsB: " << op.columnsB << std::endl;
        
        ma->setRowsUsed(op.rowsA);
        ma->setColumnsUsed(op.columnsAorRowsB);
        
        mb->setRowsUsed(op.columnsAorRowsB);
        mb->setColumnsUsed(op.columnsB);

        if (printMatrix)
        {
            std::cout << "A-> Rows: " << ma->getRowsReal() << ", Columns: " << ma->getColumnsReal() << ", Matriz A:" << std::endl;
            MatrixUtilities<Toperation>::printMatrix(ma->getRowsReal(), ma->getColumnsReal(), ma->getHostMatrix());
            std::cout << "B-> Rows: " << mb->getRowsReal() << ", Columns: " << mb->getColumnsReal() << ", Matriz B:" << std::endl;
            MatrixUtilities<Toperation>::printMatrix(mb->getRowsReal(), mb->getColumnsReal(), mb->getHostMatrix());
        }
        
        setCommOperation(op.gpuSize);
        ma->setBlockAndMeshSize(op.meshRowSize,op.meshColumnSize,op.blockRowSizeA,op.blockColumnSizeA);
        mb->setBlockAndMeshSize(op.meshRowSize,op.meshColumnSize,op.blockRowSizeB,op.blockColumnSizeB);
        ma->distributeMatrixIntoGpus();
        mb->distributeMatrixIntoGpus();
        ma->waitAllStreamsOfAllWorkers();
        mb->waitAllStreamsOfAllWorkers();
        MatrixUtilitiesCuda<Toperation>::cudaDebugMatricesLocalDifferentGpuWorkers(gpuSizeOperationWorld,op.blockRowSizeA,op.blockColumnSizeA,ma->getGpuWorkers());

        Toperation *matrixLocalC;
        //W.I.P CREO QUE ESTE NEW IRA DENTRO DEL SUMA
        mc=new MatrixMain<Toperation>(this,"C",ma->getRowsReal(), mb->getColumnsReal());
        mc->setRowsUsed(ma->getRowsUsed());
        mc->setColumnsUsed(mb->getColumnsUsed());
        std::cout<<"Llegamos sin problemas"<<std::endl;
    }


}


template class NcclMultiplicationEnvironment<double>;
