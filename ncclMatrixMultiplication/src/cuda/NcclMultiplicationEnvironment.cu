#include "hip/hip_runtime.h"
#include "NcclMultiplicationEnvironment.cuh"


template <class Toperation>
NcclMultiplicationEnvironment<Toperation>::NcclMultiplicationEnvironment(int gpuSizeWorld,int gpuRoot,OperationType opType)
{
    this->gpuSizeOperationWorld=-1;
    this->gpuRoot=gpuRoot;

    CUDACHECK(hipGetDeviceCount(&gpuSizeSystem));

    if(gpuSizeWorld!=-1)
    {
        this->gpuSizeWorld=gpuSizeWorld;
    }else
    {
        this->gpuSizeWorld=gpuSizeSystem;
    }
    this->opType=opType;
    if(opType==MultDouble)
    {
        basicOperationType=ncclDouble;
    }else
    {
        basicOperationType=ncclFloat;
    }
    std::cout<<"System gpus: "<<this->gpuSizeSystem<<". World gpus: "<<this->gpuSizeWorld<<std::endl;

    //Crear un cublasHandler con su stream correspondiente por cada gpu física del sistema
    int i;
    for(i=0;i<gpuSizeSystem;i++)
    {
        hipStream_t *newStream = new hipStream_t;
        hipblasHandle_t *newHandle = new hipblasHandle_t;
        CUDACHECK(hipStreamCreate(newStream));
        cublasStreams.push_back(newStream);
        CUBLASCHECK(hipblasCreate(newHandle));
		CUBLASCHECK(hipblasSetStream(*newHandle,*newStream));
    }
}

template <class Toperation>
int NcclMultiplicationEnvironment<Toperation>::getGpuSizeOperationWorld()
{
    return gpuSizeOperationWorld;
}
template <class Toperation>
int NcclMultiplicationEnvironment<Toperation>::getGpuSizeOperationSystem()
{
    return gpuSizeOperationSystem;
}
template <class Toperation>
int NcclMultiplicationEnvironment<Toperation>::getGpuSizeSystem()
{
    return gpuSizeSystem;
}
template <class Toperation>
int NcclMultiplicationEnvironment<Toperation>::getGpuSizeWorld()
{
    return gpuSizeWorld;
}
template <class Toperation>
int NcclMultiplicationEnvironment<Toperation>::getGpuRoot()
{
    return gpuRoot;
}

template <class Toperation>
void NcclMultiplicationEnvironment<Toperation>::setOrAddMatrixMain(std::string id, MatrixMain<Toperation> *matrixMainGlobal)
{
    matricesMatrixMain[id] = matrixMainGlobal;
}

template <class Toperation>//////////////////Por revisar esta implementacion
MatrixMain<Toperation> *NcclMultiplicationEnvironment<Toperation>::getMainMatrix(std::string id, bool create)
{
    auto it = matricesMatrixMain.find(id);
    if (it == matricesMatrixMain.end())
    {
        if (!create)
        {
            throw std::invalid_argument("La matriz global existe");
        }
        else
        {
            // Toperation *matrixAux = getMatrixGlobalSimplePointer(id);
            // dimensions dimensionsAux = matricesGlobalDimensions[id];
            // MatrixMain<Toperation> *res = new MatrixMain<Toperation>(std::get<0>(dimensionsAux), std::get<1>(dimensionsAux));
            // setOrAddMatrixMain(id, res);
            // return res;
        }
    }
    return it->second;
}

template <class Toperation>
void NcclMultiplicationEnvironment<Toperation>::setCommOperation(int gpuOperationSize)
{
    //Caso de que no se puede reutilizar el comunicador
    if(this->gpuSizeOperationWorld!=gpuOperationSize)
    {
        int i;
        if(this->gpuSizeOperationWorld!=-1)
        {
            for (i = 0; i < gpuSizeOperationSystem; ++i)
	        {
                NCCLCHECK(ncclCommDestroy(commOperation[i]));
	        }
        }
        
        this->gpuSizeOperationWorld=gpuOperationSize;
        
        gpuSizeOperationSystem=min(gpuSizeOperationWorld,gpuSizeSystem);
        int arrayGpuSystemCommOperation[gpuSizeOperationSystem];
        for(i=0;i<gpuSizeOperationSystem;i++)
        {
            arrayGpuSystemCommOperation[i]=i;
        }
        commOperation=new ncclComm_t[gpuSizeOperationSystem];
        NCCLCHECK(ncclCommInitAll(commOperation, gpuSizeOperationSystem, arrayGpuSystemCommOperation));
    }
}

template <class Toperation>
std::string NcclMultiplicationEnvironment<Toperation>::generateRandomCandiateId()
{
    std::string str("0123456789ABCDEFGHIJKLMNOPQRSTUVWXYZabcdefghijklmnopqrstuvwxyz");

    std::random_device rd;
    std::mt19937 generator(rd());

    std::shuffle(str.begin(), str.end(), generator);

    return str.substr(0, 8);
}

template <class Toperation>
std::string NcclMultiplicationEnvironment<Toperation>::generateRandomId()
{
    bool randomGenerated=false;
    std::string id;
    while(!randomGenerated)
    {
        id=generateRandomCandiateId();
        auto it = matricesMatrixMain.find(id);
        if (it == matricesMatrixMain.end())
        {
            randomGenerated=true;
        }
    }
    return id;
}

template <class Toperation>
void NcclMultiplicationEnvironment<Toperation>::performCalculations(std::string idA,std::string idB, std::string idC,bool printMatrix)
{
    OperationProperties op;
    MatrixMain<Toperation> *ma, *mb, *mc;
    ma=getMainMatrix(idA,false);
    mb=getMainMatrix(idB,false);

    if(!MatrixUtilities<Toperation>::canMultiply(ma->getColumnsReal(),mb->getRowsReal()))
    {
        throw std::invalid_argument("La operacion no se puede realizar porque las columnas no coinciden con las filas. Columnas: " +std::to_string(ma->getColumnsReal())+ ", Filas: "+ std::to_string(mb->getRowsReal()));
    }

    if(!ma->getIsDistributed() && !mb->getIsDistributed())
    {
        op = MatrixUtilities<double>::getMeshAndMatrixSize(ma->getRowsReal(), ma->getColumnsReal(), mb->getRowsReal(), mb->getColumnsReal(), gpuSizeWorld);
        std::cout << "NGpus: " << op.gpuSize << ", meshRowSize: " << op.meshRowSize << ", meshColumnSize: " << op.meshColumnSize << ", blockRowSizeA: " << \
            op.blockRowSizeA << ", blockColumnSizeA: " << op.blockColumnSizeA << ", blockRowSizeB: " << op.blockRowSizeB << ", blockColumnSizeB: " << \
            op.blockColumnSizeB << ", rowsA: " << op.rowsA << ", columnsAorRowsB: " << op.columnsAorRowsB << ", columnsB: " << op.columnsB << std::endl;
        
        ma->setRowsUsed(op.rowsA);
        ma->setColumnsUsed(op.columnsAorRowsB);
        
        mb->setRowsUsed(op.columnsAorRowsB);
        mb->setColumnsUsed(op.columnsB);

        if (printMatrix)
        {
            std::cout << "A-> Rows: " << ma->getRowsReal() << ", Columns: " << ma->getColumnsReal() << ", Matriz A:" << std::endl;
            MatrixUtilities<Toperation>::printMatrix(ma->getRowsReal(), ma->getColumnsReal(), ma->getHostMatrix());
            std::cout << "B-> Rows: " << mb->getRowsReal() << ", Columns: " << mb->getColumnsReal() << ", Matriz B:" << std::endl;
            MatrixUtilities<Toperation>::printMatrix(mb->getRowsReal(), mb->getColumnsReal(), mb->getHostMatrix());
        }
        
        setCommOperation(op.gpuSize);
        ma->setMatrixOperationProperties(op.meshRowSize,op.meshColumnSize,op.blockRowSizeA,op.blockColumnSizeA);
        mb->setMatrixOperationProperties(op.meshRowSize,op.meshColumnSize,op.blockRowSizeB,op.blockColumnSizeB);
        ma->distributeMatrixIntoGpus();
        mb->distributeMatrixIntoGpus();
        ma->waitAllStreamsOfAllWorkers();
        // mb->waitAllStreamsOfAllWorkers();
        MatrixUtilitiesCuda<Toperation>::cudaDebugMatricesLocalDifferentGpuWorkers(gpuSizeOperationWorld,op.blockRowSizeA,op.blockColumnSizeA,ma->getGpuWorkers());

        mc=mpiSumma(ma,mb,op.meshRowSize,op.meshColumnSize);
        //W.I.P CREO QUE ESTE NEW IRA DENTRO DEL SUMA
        // mc=new MatrixMain<Toperation>(this,"C",ma->getRowsReal(), mb->getColumnsReal());
        // mc->setRowsUsed(ma->getRowsUsed());
        // mc->setColumnsUsed(mb->getColumnsUsed());
        std::cout<<"Llegamos sin problemas"<<std::endl;
    }
}

template <class Toperation>
MatrixMain<Toperation>*  NcclMultiplicationEnvironment<Toperation>::mpiSumma(MatrixMain<Toperation>* matrixA, MatrixMain<Toperation>* matrixB, int meshRowsSize, int meshColumnsSize)
{
    int i;
    int rowsA = matrixA->getRowsUsed();
    int columnsAorRowsB = matrixA->getColumnsUsed();
    int columnsB = matrixB->getColumnsUsed();
    int blockSizeA = matrixA->getBlockSize();
    int blockSizeB = matrixB->getBlockSize();
    int blockRowSizeA = matrixA->getBlockRowSize();
    int blockColumnsSizeA = matrixA->getBlockColumnSize();
    int blockColumnsSizeB = matrixB->getBlockColumnSize();
    int blockRowSizeB = matrixB->getBlockRowSize();
    //Reserva de las matrices buffer para cada gpu y conseguir a que columna y fila pertenece cada gpu. Posicion i de los vectores asociadas a esa i de gpuWorker
    std::vector<Toperation*> gpuAuxiliarMatricesA,gpuAuxiliarMatricesB,gpuAuxiliarMatricesC;
    std::vector<int> rowColors,columnColors;
    for(i=0;i<gpuSizeOperationWorld;i++)
    {
        int gpuRealId=MatrixUtilitiesCuda<Toperation>::getRealGpuId(i,gpuSizeSystem);
        CUDACHECK(hipSetDevice(gpuRealId));
        Toperation *gpuAuxA=MatrixUtilitiesCuda<Toperation>::cudaMatrixMemoryAllocation(blockRowSizeA,blockColumnsSizeA,cublasStreams[gpuRealId]);
        Toperation *gpuAuxB=MatrixUtilitiesCuda<Toperation>::cudaMatrixMemoryAllocation(blockRowSizeB,blockColumnsSizeB,cublasStreams[gpuRealId]);
        Toperation *gpuAuxC=MatrixUtilitiesCuda<Toperation>::cudaMatrixMemoryAllocation(blockRowSizeA,blockColumnsSizeB,cublasStreams[gpuRealId]);
        gpuAuxiliarMatricesA.push_back(gpuAuxA);gpuAuxiliarMatricesB.push_back(gpuAuxB);gpuAuxiliarMatricesC.push_back(gpuAuxC);
        rowColors.push_back(matrixA->calculateRowColor(i));
        columnColors.push_back(matrixA->calculateColumnColor(i));
    }
    
    //Creacion de los nuevos grupos comunicadores para hacer Broadcast de filas o columnas a los pertenecientes a la malla de misma fila o columna
    // int colGroupIndex[meshRowsSize];
    // int rowGroupIndex[meshColumnsSize];
    // for (i = 0; i < meshColumnsSize; i++)
    // {
    //     rowGroupIndex[i] = rowColor * meshColumnsSize + i;
    // }
    // for (i = 0; i < meshRowsSize; i++)
    // {
    //     colGroupIndex[i] = columnColor + i * meshColumnsSize;
    // }
    // MPI_Group_incl(groupInitial, meshColumnsSize, rowGroupIndex, &groupRow);
    // MPI_Group_incl(groupInitial, meshRowsSize, colGroupIndex, &groupColumn);
    // MPI_Comm_create(commOperation, groupRow, &commRow);
    // MPI_Comm_create(commOperation, groupColumn, &commCol);
    // //Realizacion de las operaciones matematicas
    // for (i = 0; i < meshColumnsSize; i++)
    // {
    //     if (columnColor == (i % meshColumnsSize))
    //     {
    //         memcpy(matrixAuxiliarA, matrixLocalA.getMatrixLocal(i / meshColumnsSize), blockSizeA * sizeof(Toperation));
    //     }
    //     if (rowColor == (i % meshRowsSize))
    //     {
    //         memcpy(matrixAuxiliarB, matrixLocalB.getMatrixLocal(i / meshRowsSize), blockSizeB * sizeof(Toperation));
    //     }
    //     MPI_Bcast(matrixAuxiliarA, blockSizeA, basicOperationType, (i % meshColumnsSize), commRow);
    //     MPI_Bcast(matrixAuxiliarB, blockSizeB, basicOperationType, (i % meshRowsSize), commCol);
    //     // MatrixUtilities<Toperation>::Multiplicacion(blockRowSizeA, blockRowSizeB, blockColumnsSizeB, matrixAuxiliarA, matrixAuxiliarB, matrixLocalC);
    //     MatrixUtilities<Toperation>::matrixBlasMultiplication(blockRowSizeA, blockRowSizeB, blockColumnsSizeB, matrixAuxiliarA, matrixAuxiliarB, matrixLocalC);
    // }
    // //Liberacion de las matrices auxiliares que realizaban computo
    // MatrixUtilities<Toperation>::matrixFree(matrixAuxiliarA);
    // MatrixUtilities<Toperation>::matrixFree(matrixAuxiliarB);

    // return matrixLocalC;
    return nullptr;
    
}


template class NcclMultiplicationEnvironment<double>;
