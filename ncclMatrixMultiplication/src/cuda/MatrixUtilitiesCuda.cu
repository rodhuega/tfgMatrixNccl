#include "hip/hip_runtime.h"
#include "MatrixUtilitiesCuda.cuh"


__global__ void
cudaPrintMatrix(int rows,int columns,double* matrix)
{
	for(int i =0;i<rows;i++)
	{
		for(int j=0;j<columns;j++)
		{
			printf("%.2lf\t",matrix[IDX2C(i,j,rows)]);
		}
		printf("\n");
	}
}


void MatrixUtilitiesCuda::cudaPrintMatrixCall(int rows,int columns,double* matrix)
{
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    printf("HOLA\n");
    double* prueba;
    hipMalloc ((void**)&prueba, rows*columns*sizeof(double));
    hipMemcpy(prueba,matrix,rows*columns*sizeof(double),hipMemcpyHostToDevice);
    cudaPrintMatrix<<<1,1,1>>>(rows,columns,prueba);
    hipDeviceSynchronize();
}