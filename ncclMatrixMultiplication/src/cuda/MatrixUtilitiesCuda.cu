#include "hip/hip_runtime.h"
#include "MatrixUtilitiesCuda.cuh"
#include "ErrorCheckingCuda.cuh"


__global__ void
cudaPrintMatrix(int rows,int columns,double* matrix)
{
	for(int i =0;i<rows;i++)
	{
		for(int j=0;j<columns;j++)
		{
			printf("%.2f\t",matrix[IDX2C(i,j,rows)]);
		}
		printf("\n");
	}
}

template <class Toperation>
int MatrixUtilitiesCuda<Toperation>::matrixCalculateIndex(int rowSize, int columnSize, int rowIndex, int columnIndex)
{
    return IDX2C(rowIndex,columnIndex,rowSize);
    // return columnSize * rowIndex + columnIndex;
}
template <class Toperation>
Toperation* MatrixUtilitiesCuda<Toperation>::cudaMatrixMemoryAllocation(int rows, int columns,hipStream_t *stream)
{
    Toperation* newMatrix;
    CUDACHECK(hipMalloc ((void**)&newMatrix,rows*columns*sizeof(double)));
    CUDACHECK(hipMemsetAsync(newMatrix, 0, sizeof(double)*rows*columns,*stream));
    return newMatrix;
}

template <class Toperation>
int MatrixUtilitiesCuda<Toperation>::getRealGpuId(int gpuRankOperation,int gpuSizeSystem)
{
    return gpuRankOperation%gpuSizeSystem;
}

template <class Toperation>
void MatrixUtilitiesCuda<Toperation>::cudaPrintOneMatrixCall(int rows,int columns,Toperation* matrix)
{
    cudaPrintMatrix<<<1,1,1>>>(rows,columns,(double*)matrix);
    CUDACHECK(hipDeviceSynchronize());
}

template <class Toperation>
void MatrixUtilitiesCuda<Toperation>::cudaDebugMatrixDifferentGpus(int gpuRank, int rows, int columns, Toperation *M, std::string extraMessage)
{
    usleep(gpuRank * 1000);
    std::cout << "Parte del gpuWorker: " << gpuRank << " " << extraMessage << std::endl;
    MatrixUtilitiesCuda::cudaPrintOneMatrixCall(rows, columns, M);
}

template <class Toperation>
void MatrixUtilitiesCuda<Toperation>::cudaDebugMatricesLocalDifferentGpuWorkers(int gpuSize, int rows, int columns, std::vector<GpuWorker<Toperation>*> gpuWorkers)
{
    unsigned int gpuRank,j;
    for (gpuRank = 0; gpuRank < gpuWorkers.size(); gpuRank++)
    {
        for(j=0;j<gpuWorkers[gpuRank]->getMatricesLocal().size();j++)
        {
            //W.I.P CREO QUE EL CALCULO DEL TOSTRING ESTA MAL
            std::string msg =" Id gpu real: "+std::to_string(gpuWorkers[gpuRank]->getGpuRankSystem()) +" Matriz local: " + std::to_string((gpuRank + (j * gpuSize)));
            MatrixUtilitiesCuda::cudaDebugMatrixDifferentGpus(gpuRank, rows, columns, gpuWorkers[gpuRank]->getMatricesLocal()[j], msg);
        }
    }
}

template <class Toperation>
void MatrixUtilitiesCuda<Toperation>::matrixCublasMultiplication(hipblasHandle_t* handler,int rowsA, int columnsAorRowsB, int columnsB, Toperation *A, Toperation *B, Toperation *C)
{
    double alfa=1;double beta=1.0;
    // cblas_dgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, rowsA, columnsB, columnsAorRowsB, 1.0, (double*)A, columnsAorRowsB, (double*)B, columnsB, 1.0, (double*)C, columnsB);
    CUBLASCHECK(hipblasDgemm(*handler, HIPBLAS_OP_N, HIPBLAS_OP_N, rowsA, columnsB, columnsAorRowsB, &alfa, (double*)A, columnsAorRowsB, (double*)B, columnsB, &beta, (double*)C, rowsA));

}


template class MatrixUtilitiesCuda<double>;