#include "hip/hip_runtime.h"
#include "MatrixUtilitiesCuda.cuh"
#include "ErrorCheckingCuda.cuh"


__global__ void
cudaPrintMatrix(int rows,int columns,double* matrix)
{
	for(int i =0;i<rows;i++)
	{
		for(int j=0;j<columns;j++)
		{
			printf("%.2lf\t",matrix[IDX2C(i,j,rows)]);
		}
		printf("\n");
	}
}

template <class Toperation>
int MatrixUtilitiesCuda<Toperation>::matrixCalculateIndex(int rowSize, int columnSize, int rowIndex, int columnIndex)
{
    return IDX2C(rowIndex,columnIndex,rowSize);
    // return columnSize * rowIndex + columnIndex;
}
template <class Toperation>
Toperation* MatrixUtilitiesCuda<Toperation>::cudaMatrixMemoryAllocation(int rows, int columns)
{
    Toperation* newMatrix;
    CUDACHECK(hipMalloc ((void**)&newMatrix,rows*columns*sizeof(double)));
    CUDACHECK(hipMemsetAsync(newMatrix, 0, sizeof(double)*rows*columns,0));
    return newMatrix;
}

template <class Toperation>
int MatrixUtilitiesCuda<Toperation>::getRealGpuId(int gpuRankOperation,int gpuSizeSystem)
{
    return gpuRankOperation%gpuSizeSystem;
}

template <class Toperation>
void MatrixUtilitiesCuda<Toperation>::cudaPrintMatrixCall(int rows,int columns,double* matrix)
{
    cudaPrintMatrix<<<1,1,1>>>(rows,columns,matrix);
}

template class MatrixUtilitiesCuda<double>;