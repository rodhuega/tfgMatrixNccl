#include "hip/hip_runtime.h"
#include "MatrixUtilitiesCuda.cuh"
#include "ErrorCheckingCuda.cuh"


__global__ void
cudaPrintMatrix(int rows,int columns,double* matrix)
{
	for(int i =0;i<rows;i++)
	{
		for(int j=0;j<columns;j++)
		{
			printf("%.2lf\t",matrix[IDX2C(i,j,rows)]);
		}
		printf("\n");
	}
}

template <class Toperation>
int MatrixUtilitiesCuda<Toperation>::matrixCalculateIndex(int rowSize, int columnSize, int rowIndex, int columnIndex)
{
    return IDX2C(rowIndex,columnIndex,rowSize);
    // return columnSize * rowIndex + columnIndex;
}
template <class Toperation>
Toperation* MatrixUtilitiesCuda<Toperation>::cudaMatrixMemoryAllocation(int rows, int columns)
{
    // CUDACHECK(hipEventRecord(startMalloc1[i]));
    // CUDACHECK(hipMalloc ((void**)&gpusInfo[i]->matrixDeviceA,rowsA*columnsA*sizeof(double)));
    // CUDACHECK(hipEventRecord(stopMalloc1[i]));
    // CUDACHECK(hipEventRecord(startMemSet1[i],gpusInfo[i]->streams[0]));
    // CUDACHECK(hipMemsetAsync(gpusInfo[i]->matrixDeviceA, 0, sizeof(double)*rowsA*columnsA,gpusInfo[i]->streams[0]));
    // CUDACHECK(hipEventRecord(stopMemSet1[i],gpusInfo[i]->streams[0]));

}

template <class Toperation>
int MatrixUtilitiesCuda<Toperation>::getRealGpuId(int gpuRankOperation,int gpuSizeSystem)
{
    return gpuRankOperation%gpuSizeSystem;
}

template <class Toperation>
void MatrixUtilitiesCuda<Toperation>::cudaPrintMatrixCall(int rows,int columns,double* matrix)
{
    hipblasHandle_t handle;
    CUBLASCHECK(hipblasCreate(&handle));
    printf("HOLA\n");
    double* prueba;
    CUDACHECK(hipMalloc ((void**)&prueba, rows*columns*sizeof(double)));
    CUDACHECK(hipMemcpy(prueba,matrix,rows*columns*sizeof(double),hipMemcpyHostToDevice));
    cudaPrintMatrix<<<1,1,1>>>(rows,columns,prueba);
    CUDACHECK(hipDeviceSynchronize());
}

template class MatrixUtilitiesCuda<double>;