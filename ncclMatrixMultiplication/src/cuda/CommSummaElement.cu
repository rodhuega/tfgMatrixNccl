#include "CommSummaElement.cuh"


CommSummaElement::CommSummaElement(int idGpuLogic,int idGpuPhysical,int rowColor,int columnColor)
{
    this->idGpuLogic=idGpuLogic;
    this->idGpuPhysical=idGpuPhysical;
    this->rowColor=rowColor;
    this->columnColor=columnColor;
    CUDACHECK(hipSetDevice(idGpuPhysical));
    streamRow= new hipStream_t;streamColumn= new hipStream_t;streamRowMySelf= new hipStream_t;streamColumnMySelf= new hipStream_t;
    CUDACHECK(hipStreamCreate(streamRow));
    CUDACHECK(hipStreamCreate(streamColumn));;
    CUDACHECK(hipStreamCreate(streamRowMySelf));
    CUDACHECK(hipStreamCreate(streamColumnMySelf));
}

CommSummaElement::~CommSummaElement()
{
    if(idGpuLogic==idGpuPhysical)
    {
        NCCLCHECK(ncclCommDestroy(commRow));
        NCCLCHECK(ncclCommDestroy(commColumn));
        if(streamRow!=nullptr)
        {
            CUDACHECK(hipStreamDestroy(*streamRow));
        }
        if(streamColumn!=nullptr)
        {
            CUDACHECK(hipStreamDestroy(*streamColumn));
        }
    }
}

int CommSummaElement::getIdLogic()
{
    return idGpuLogic;
}


int CommSummaElement::getIdPhysical()
{
    return idGpuPhysical;
}


std::vector<int>  CommSummaElement::getRanksCommsRowsPhysical()
{
    return ranksCommsRowsPhysical;
}

std::vector<int>  CommSummaElement::getRanksCommsColumnsPhysical()
{
    return ranksCommsColumnsPhysical;
}

int CommSummaElement::getRankCommRowLogic()
{
    return rankCommRowLogic;
}

int CommSummaElement::getRankCommColumnLogic()
{
    return rankCommColumnLogic;
}

int CommSummaElement::getRowColor()
{
    return rowColor;
}

int CommSummaElement::getColumnColor()
{
    return columnColor;
}

std::vector<std::vector<int>> CommSummaElement::getRowDevices()
{
    return rowDevices;
}
std::vector<std::vector<int>> CommSummaElement::getColumnDevices()
{
    return columnDevices;
}

ncclComm_t CommSummaElement::getCommRow()
{
    return commRow;
}

ncclComm_t CommSummaElement::getCommColumn()
{
    return commColumn;
}

ncclComm_t CommSummaElement::getCommRowMySelf()
{
    return commRowMySelf;
}

ncclComm_t CommSummaElement::getCommColumnMySelf()
{
    return commColumnMySelf;
}

hipStream_t* CommSummaElement::getStreamRow()
{
    return streamRow;
}

hipStream_t* CommSummaElement::getStreamColumn()
{
    return streamColumn;
}

hipStream_t* CommSummaElement::getStreamRowMySelf()
{
    return streamRowMySelf;
}

hipStream_t* CommSummaElement::getStreamColumnMySelf()
{
    return streamColumnMySelf;
}

void CommSummaElement::addRankCommRowPhysical(int rankCommRowPhysical)
{
    this->ranksCommsRowsPhysical.push_back(rankCommRowPhysical);
}

void CommSummaElement::addRankCommColumnPhysical(int rankCommColumnPhysical)
{
    this->ranksCommsColumnsPhysical.push_back(rankCommColumnPhysical);
}

void CommSummaElement::setRankCommRowLogic(int rankCommRowLogic)
{
    this->rankCommRowLogic=rankCommRowLogic;
}

void CommSummaElement::setRankCommColumnLogic(int rankCommColumnLogic)
{
    this->rankCommColumnLogic=rankCommColumnLogic;
}

void CommSummaElement::setRowDevices(std::vector<std::vector<int>> rowDevices)
{
    this->rowDevices=rowDevices;
}
        
void CommSummaElement::setColumnDevices(std::vector<std::vector<int>> columnDevices)
{
    this->columnDevices=columnDevices;
}

void CommSummaElement::setCommRow(ncclComm_t commRow)
{
    this->commRow=commRow;
}

void CommSummaElement::setCommColumn(ncclComm_t commColumn)
{
    this->commColumn=commColumn;
}

void CommSummaElement::setCommRowMySelf(ncclComm_t commRowMySelf)
{
    this->commRowMySelf=commRowMySelf;
}

void CommSummaElement::setCommColumnMySelf(ncclComm_t commColumnMySelf)
{
    this->commColumnMySelf=commColumnMySelf;
}

void CommSummaElement::setStreamRow(hipStream_t* streamRow)
{
    this->streamRow=streamRow;
}

void CommSummaElement::setStreamColumn(hipStream_t* streamColumn)
{
    this->streamColumn=streamColumn;
}