#include "CommSummaElement.cuh"


CommSummaElement::CommSummaElement(int idGpuLogic,int idGpuPhysical,int rowColor,int columnColor)
{
    this->idGpuLogic=idGpuLogic;
    this->idGpuPhysical=idGpuPhysical;
    this->rowColor=rowColor;
    this->columnColor=columnColor;
    this->lastRowMySelf=0;
    this->lastColumnMySelf=lastColumnMySelf;
    CUDACHECK(hipSetDevice(idGpuPhysical));
    streamRow= new hipStream_t;streamColumn= new hipStream_t;
    CUDACHECK(hipStreamCreate(streamRow));
    CUDACHECK(hipStreamCreate(streamColumn));
}

CommSummaElement::~CommSummaElement()
{
    if(idGpuLogic==idGpuPhysical)
    {
        NCCLCHECK(ncclCommDestroy(commRow));
        NCCLCHECK(ncclCommDestroy(commColumn));
        if(streamRow!=nullptr)
        {
            CUDACHECK(hipStreamDestroy(*streamRow));
        }
        if(streamColumn!=nullptr)
        {
            CUDACHECK(hipStreamDestroy(*streamColumn));
        }
    }
}

int CommSummaElement::getIdLogic()
{
    return idGpuLogic;
}


int CommSummaElement::getIdPhysical()
{
    return idGpuPhysical;
}


std::vector<int>  CommSummaElement::getRanksCommsRowsPhysical()
{
    return ranksCommsRowsPhysical;
}

std::vector<int>  CommSummaElement::getRanksCommsColumnsPhysical()
{
    return ranksCommsColumnsPhysical;
}

int CommSummaElement::getRankCommRowLogic()
{
    return rankCommRowLogic;
}

int CommSummaElement::getRankCommColumnLogic()
{
    return rankCommColumnLogic;
}

int CommSummaElement::getRowColor()
{
    return rowColor;
}

int CommSummaElement::getColumnColor()
{
    return columnColor;
}

std::vector<std::vector<int>> CommSummaElement::getRowDevices()
{
    return rowDevices;
}
std::vector<std::vector<int>> CommSummaElement::getColumnDevices()
{
    return columnDevices;
}

ncclComm_t CommSummaElement::getCommRow()
{
    return commRow;
}

ncclComm_t CommSummaElement::getCommColumn()
{
    return commColumn;
}

ncclComm_t CommSummaElement::getCommRowMySelf()
{
    return commsRowsMySelf[lastRowMySelf++];
}

ncclComm_t CommSummaElement::getCommColumnMySelf()
{
    return commsColumnsMySelf[lastColumnMySelf++];
}

hipStream_t* CommSummaElement::getStreamRow()
{
    return streamRow;
}

hipStream_t* CommSummaElement::getStreamColumn()
{
    return streamColumn;
}

hipStream_t* CommSummaElement::getStreamRowMySelf()
{
    return streamsRowsMySelf[lastRowMySelf];
}

hipStream_t* CommSummaElement::getStreamColumnMySelf()
{
    return streamsColumnsMySelf[lastColumnMySelf];
}

void CommSummaElement::addRankCommRowPhysical(int rankCommRowPhysical)
{
    this->ranksCommsRowsPhysical.push_back(rankCommRowPhysical);
}

void CommSummaElement::addRankCommColumnPhysical(int rankCommColumnPhysical)
{
    this->ranksCommsColumnsPhysical.push_back(rankCommColumnPhysical);
}

void CommSummaElement::setRankCommRowLogic(int rankCommRowLogic)
{
    this->rankCommRowLogic=rankCommRowLogic;
}

void CommSummaElement::setRankCommColumnLogic(int rankCommColumnLogic)
{
    this->rankCommColumnLogic=rankCommColumnLogic;
}

void CommSummaElement::setRowDevices(std::vector<std::vector<int>> rowDevices)
{
    this->rowDevices=rowDevices;
}
        
void CommSummaElement::setColumnDevices(std::vector<std::vector<int>> columnDevices)
{
    this->columnDevices=columnDevices;
}

void CommSummaElement::setCommRow(ncclComm_t commRow)
{
    this->commRow=commRow;
}

void CommSummaElement::setCommColumn(ncclComm_t commColumn)
{
    this->commColumn=commColumn;
}

void CommSummaElement::addCommRowMySelf(ncclComm_t commRowMySelf)
{
    this->commsRowsMySelf.push_back(commRowMySelf);
    hipStream_t * newStream=new hipStream_t;
    CUDACHECK(hipStreamCreate(newStream));
    streamsRowsMySelf.push_back(newStream);
}

void CommSummaElement::addCommColumnMySelf(ncclComm_t commColumnMySelf)
{
    this->commsColumnsMySelf.push_back(commColumnMySelf);
    hipStream_t * newStream=new hipStream_t;
    CUDACHECK(hipStreamCreate(newStream));
    streamsColumnsMySelf.push_back(newStream);
}

void CommSummaElement::setStreamRow(hipStream_t* streamRow)
{
    this->streamRow=streamRow;
}

void CommSummaElement::setStreamColumn(hipStream_t* streamColumn)
{
    this->streamColumn=streamColumn;
}

void CommSummaElement::waitStreams()
{
    lastColumnMySelf=0;
    lastRowMySelf=0;
}