#include "hip/hip_runtime.h"
#include "MatrixMain.cuh"

template <class Toperation>
MatrixMain<Toperation>::MatrixMain(NcclMultiplicationEnvironment<Toperation>* ncclMultEnv,std::string id,int rows,int columns)
{
    this->ncclMultEnv=ncclMultEnv;
    this->id=id;
    this->rowsReal=rows;
    this->columnsReal=columns;
    this->isMatrixHostHere=false;
    this->isDistributed=false;
    this->deleteMatrixHostAtDestroyment=false;
    this->deleteObjectAtDestroyment=true;
    this->hostMatrix=nullptr;
    this->alphaGemm=1;
    if(id=="")
    {
        this->id=this->ncclMultEnv->generateRandomId();
    }
}

template <class Toperation>
MatrixMain<Toperation>::MatrixMain(NcclMultiplicationEnvironment<Toperation>* ncclMultEnv,std::string id,int rows,int columns,Toperation* matrix):MatrixMain(ncclMultEnv,id,rows,columns)
{
    this->ncclMultEnv=ncclMultEnv;
    this->hostMatrix=matrix;
    this->isMatrixHostHere=true;
}

template <class Toperation>
MatrixMain<Toperation>::MatrixMain(const MatrixMain<Toperation> &maMain)
{
    this->id=this->ncclMultEnv->generateRandomId();
    assignationToActualObject(maMain,false,true);
}


template <class Toperation>
MatrixMain<Toperation>::~MatrixMain()
{
    if(deleteObjectAtDestroyment)
    {
        if(isMatrixHostHere && deleteMatrixHostAtDestroyment)
        {
            MatrixUtilities<Toperation>::matrixFree(hostMatrix);
        }
        deleteGpuWorkers();
    }
}

template <class Toperation>
std::string  MatrixMain<Toperation>::getId()
{
    return id;
}
template <class Toperation>
int MatrixMain<Toperation>::getRowsReal()
{
    return rowsReal;
}

template <class Toperation>
int MatrixMain<Toperation>::getColumnsReal()
{
    return columnsReal;
}

template <class Toperation>
int MatrixMain<Toperation>::getRowsUsed()
{
    return rowsUsed;
}

template <class Toperation>
int MatrixMain<Toperation>::getColumnsUsed()
{
    return columnsUsed;
}

template <class Toperation>
bool MatrixMain<Toperation>::getIsDistributed()
{
    return isDistributed;
}

template <class Toperation>
int MatrixMain<Toperation>::getBlockSize()
{
    return blockSize;
}

template <class Toperation>
int MatrixMain<Toperation>::getBlockRowSize()
{
    return blockRowSize;
}

template <class Toperation>
int MatrixMain<Toperation>::getBlockColumnSize()
{
    return blockColumnSize;
}

template <class Toperation>
int MatrixMain<Toperation>::getMeshColumnSize()
{
    return meshColumnSize;
}

template <class Toperation>
int MatrixMain<Toperation>::getMeshRowSize()
{
    return meshRowSize;
}

template <class Toperation>
Toperation MatrixMain<Toperation>::getAlphaGemm()
{
    return alphaGemm;
}

template <class Toperation>
bool MatrixMain<Toperation>::getIsMatrixHostHere()
{
    return isMatrixHostHere;
}

template <class Toperation>
bool MatrixMain<Toperation>::getDeleteMatrixHostAtDestroyment()
{
    return deleteMatrixHostAtDestroyment;
}

template <class Toperation>
Toperation *MatrixMain<Toperation>::getHostMatrix()
{
    if(!isMatrixHostHere)
    {
        recoverMatrixToHost();
    }
    return hostMatrix;
}
template <class Toperation>
std::vector<GpuWorker<Toperation>*> MatrixMain<Toperation>::getGpuWorkers()
{
    return gpuWorkers;
}

template <class Toperation>
void MatrixMain<Toperation>::setId(std::string id)
{
    this->id=id;
}

template <class Toperation>
void MatrixMain<Toperation>::setRowsUsed(int rowsUsed)
{
    this->rowsUsed = rowsUsed;
}

template <class Toperation>
void MatrixMain<Toperation>::setColumnsUsed(int columnsUsed)
{
    this->columnsUsed = columnsUsed;
}

template <class Toperation>
void MatrixMain<Toperation>::setAlphaGemm(Toperation alphaGemm)
{
    this->alphaGemm = alphaGemm;
}

template <class Toperation>
void MatrixMain<Toperation>::setIsDistributed(bool isDistributed)
{
    this->isDistributed = isDistributed;
}

template <class Toperation>
void MatrixMain<Toperation>::setIsMatrixHostHere(bool isMatrixHostHere)
{
    this->isMatrixHostHere = isMatrixHostHere;
    if(!isMatrixHostHere && hostMatrix!=nullptr)
    {
        MatrixUtilities<Toperation>::matrixFree(hostMatrix);
    }
}

template <class Toperation>
void MatrixMain<Toperation>::setDeleteMatrixHostAtDestroyment(bool deleteMatrixHostAtDestroyment)
{
    this->deleteMatrixHostAtDestroyment=deleteMatrixHostAtDestroyment;
}


template <class Toperation>
void MatrixMain<Toperation>::setMatrixOperationProperties(int meshRowSize, int meshColumnSize, int blockRowSize, int blockColumnSize)
{
    this->isDistributed=false;
    gpuWorkers.clear();
    this->meshRowSize=meshRowSize;
    this->meshColumnSize=meshColumnSize;
    this->blockRowSize=blockRowSize;
    this->blockColumnSize=blockColumnSize;
    this->numberOfRowBlocks = ceil(this->rowsUsed / this->blockRowSize);
    this->numberOfColumnBlocks = ceil(this->columnsUsed / this->blockColumnSize);
    this->numberOfTotalBlocks = this->numberOfRowBlocks * this->numberOfColumnBlocks;
    this->blockSize = this->blockRowSize * this->blockColumnSize;
    blocksInitialPosition.resize(numberOfTotalBlocks);
    blocksInitialPositionDiagonal.resize(numberOfTotalBlocks);
    int i, posColumnBelong, posRowBelong,indexBlock,gpuRealId,actualIndexDiagonal=0;
    for (i = 0,indexBlock=0; i < numberOfTotalBlocks; i++)
    {
        posColumnBelong = (i % numberOfColumnBlocks) * rowsReal * blockColumnSize;
        posRowBelong = (i / numberOfColumnBlocks) * blockRowSize;
        blocksInitialPosition[i]=(posColumnBelong + posRowBelong);
        if(actualIndexDiagonal>=blocksInitialPosition[i] && actualIndexDiagonal<(blocksInitialPosition[i]+blockRowSize))
        {
            blocksInitialPositionDiagonal[i]=actualIndexDiagonal;
            actualIndexDiagonal+=rowsReal*blockColumnSize+min(blockRowSize,blockColumnSize);
        }else
        {
            blocksInitialPositionDiagonal[i]=-1;
        }
        //W.I.P CREo QUE SOBRA este if
        //Debido a ColumnMajorOrder corrijo al indice del bloque que pertenece para una correcta formación de la malla.
        indexBlock=(indexBlock+numberOfColumnBlocks);
        if(indexBlock>=numberOfTotalBlocks){
            indexBlock%=(numberOfTotalBlocks-1);
        }
        //Creacion de los gpuWorkers y su primer bloque
        if(i<ncclMultEnv->getGpuSizeOperationWorld())
        {
            gpuRealId=MatrixUtilitiesCuda<Toperation>::getRealGpuId(i,ncclMultEnv->getGpuSizeSystem());
            CUDACHECK(hipSetDevice(gpuRealId));
            GpuWorker<Toperation> *gpuW= new GpuWorker<Toperation>(i,gpuRealId,this);
            gpuWorkers.push_back(gpuW);
            hipStream_t *newStream = new hipStream_t;
            CUDACHECK(hipStreamCreate(newStream));
            gpuWorkers[i]->addStream(newStream);
            Toperation *newMatrix=MatrixUtilitiesCuda<Toperation>::cudaMatrixMemoryAllocation(blockRowSize,blockColumnSize,newStream);
            gpuWorkers[i]->addMatrixLocal(newMatrix);
        }
    }
}

template <class Toperation>
int MatrixMain<Toperation>::calculateRowColor(int gpuRank)
{
    return gpuRank / numberOfColumnBlocks;
}

template <class Toperation>
int MatrixMain<Toperation>::calculateColumnColor(int gpuRank)
{
    return gpuRank % numberOfColumnBlocks;
}

template <class Toperation>
int MatrixMain<Toperation>::calculateBlockDimensionToCopy(int color, int meshDimensionSize, int blockDimenensionSize, int dimensionUsed, int dimensionReal)
{
    return (color != (meshDimensionSize - 1)) ? blockDimenensionSize : (blockDimenensionSize - (dimensionUsed - dimensionReal));
}

template <class Toperation>
void MatrixMain<Toperation>::waitAllStreamsOfAllWorkers()
{
    int i;
    for(i=0;i<gpuWorkers.size();i++)
    {
        CUDACHECK(hipSetDevice(gpuWorkers[i]->getGpuRankSystem()));
        gpuWorkers[i]->waitAllStreams();
    }
}

template <class Toperation>
void MatrixMain<Toperation>::distributeMatrixIntoGpus()
{
    if(!isDistributed)
    {
        if(!isMatrixHostHere)
        {
            throw std::invalid_argument("No existe matriz en el host, asi que no se puede distribuir");
        }
        int i,j,k,blockColumnSizeCopy,blockRowSizeCopy;
        for(i=0;i<ncclMultEnv->getGpuSizeOperationWorld()&&i<numberOfTotalBlocks;i++)
        {
            CUDACHECK(hipSetDevice(gpuWorkers[i]->getGpuRankSystem()));
            for(j=i;j<numberOfTotalBlocks;j+=ncclMultEnv->getGpuSizeOperationWorld())
            {
                Toperation *newMatrix;
                hipStream_t *newStream;
                if(j!=i)//El primer bloque ya estaba creado de la llamada a setMatrixOperationProperties
                {
                    newStream = new hipStream_t;
                    CUDACHECK(hipStreamCreate(newStream));
                    gpuWorkers[i]->addStream(newStream);
                    newMatrix=MatrixUtilitiesCuda<Toperation>::cudaMatrixMemoryAllocation(blockRowSize,blockColumnSize,newStream);
                    gpuWorkers[i]->addMatrixLocal(newMatrix);
                }else 
                {
                    newStream=gpuWorkers[i]->getStream(0);
                    newMatrix=gpuWorkers[i]->getMatrixLocal(0);
                }
                blockColumnSizeCopy = calculateBlockDimensionToCopy(calculateColumnColor(i), numberOfColumnBlocks, blockColumnSize, columnsUsed, columnsReal);
                blockRowSizeCopy = calculateBlockDimensionToCopy(calculateRowColor(i), numberOfRowBlocks, blockRowSize, rowsUsed, rowsReal);
                for(k=0;k<blockColumnSizeCopy;k++)
                {
                    CUDACHECK(hipMemcpyAsync(&newMatrix[k*blockRowSize],&hostMatrix[blocksInitialPosition[j]+k*rowsReal],blockRowSizeCopy*sizeof(Toperation),hipMemcpyHostToDevice,*newStream));
                }
            }
    }
    setIsDistributed(true);
    }
}

template <class Toperation>
void MatrixMain<Toperation>::recoverMatrixToHost()
{
    if(!isMatrixHostHere)
    {
        if(gpuWorkers.size()==0)
        {
            throw std::invalid_argument("La matriz no se encuentra distribuida, asi que no se puede recuperar.");
        }
        int i,j,k,blockColumnSizeCopy,blockRowSizeCopy,matrixLocalIndex;
        hostMatrix=MatrixUtilities<Toperation>::matrixMemoryAllocation(rowsReal,columnsReal);
        for(i=0;i<ncclMultEnv->getGpuSizeOperationWorld()&&i<numberOfTotalBlocks;i++)
        {
            CUDACHECK(hipSetDevice(gpuWorkers[i]->getGpuRankSystem()));
            for(j=i,matrixLocalIndex=0;j<numberOfTotalBlocks;j+=ncclMultEnv->getGpuSizeOperationWorld(),matrixLocalIndex++)
            {
                Toperation *newMatrix;
                hipStream_t *newStream;
                newStream=gpuWorkers[i]->getStream(matrixLocalIndex);
                newMatrix=gpuWorkers[i]->getMatrixLocal(matrixLocalIndex);
                blockColumnSizeCopy = calculateBlockDimensionToCopy(calculateColumnColor(i), numberOfColumnBlocks, blockColumnSize, columnsUsed, columnsReal);
                blockRowSizeCopy = calculateBlockDimensionToCopy(calculateRowColor(i), numberOfRowBlocks, blockRowSize, rowsUsed, rowsReal);
                for(k=0;k<blockColumnSizeCopy;k++)
                {
                    CUDACHECK(hipMemcpyAsync(&hostMatrix[blocksInitialPosition[j]+k*rowsReal],&newMatrix[k*blockRowSize],blockRowSizeCopy*sizeof(Toperation),hipMemcpyDeviceToHost,*newStream));
                }
            }
        }
        waitAllStreamsOfAllWorkers();
        setIsMatrixHostHere(true);
    }
}

template <class Toperation>
void MatrixMain<Toperation>::deleteGpuWorkers()
{
    int i;
    for(i=0;i<this->gpuWorkers.size();i++)
    {
         delete this->gpuWorkers[i];
    }
    this->gpuWorkers.clear();
}

template <class Toperation>
void MatrixMain<Toperation>::assignationToActualObject(const MatrixMain<Toperation>& B,bool sameId,bool deepCopy)
{
    deleteGpuWorkers();
    this->ncclMultEnv=B.ncclMultEnv;
    this->deleteObjectAtDestroyment=B.deleteObjectAtDestroyment;
    this->deleteMatrixHostAtDestroyment=B.deleteMatrixHostAtDestroyment;
    this->blocksInitialPosition=B.blocksInitialPosition;
    this->rowsReal=B.rowsReal;
    this->rowsUsed=B.rowsUsed;
    this->columnsReal=B.columnsReal;
    this->columnsUsed=B.columnsUsed;
    this->isDistributed=B.isDistributed;
    this->isMatrixHostHere=B.isMatrixHostHere;
    this->blockRowSize=B.blockRowSize;
    this->blockColumnSize=B.blockColumnSize;
    this->blockSize=B.blockSize;    
    this->meshRowSize=B.meshRowSize;
    this->meshColumnSize=B.meshColumnSize;
    this->numberOfRowBlocks=B.numberOfRowBlocks;
    this->numberOfColumnBlocks=B.numberOfColumnBlocks;
    this->numberOfTotalBlocks=B.numberOfTotalBlocks;
    if(sameId)
    {
        this->id=B.id;
    }   
    if(deepCopy)
    {
        if(isMatrixHostHere)
        {
            this->hostMatrix=MatrixUtilities<Toperation>::matrixMemoryAllocation(this->rowsReal, this->columnsReal);
            memcpy(this->hostMatrix,B.hostMatrix,sizeof(Toperation)*this->rowsReal*this->columnsReal);
        }else
        {
            this->hostMatrix=nullptr;
        }

        if(isDistributed)
        {
            int i;
            GpuWorker<Toperation>* aux;
            for(i=0;i<B.gpuWorkers.size();i++)
            {
                aux= new GpuWorker<Toperation>(*B.gpuWorkers[i]);
                this->gpuWorkers.push_back(aux);
            }
            this->waitAllStreamsOfAllWorkers();
        }

    }else
    {
        this->hostMatrix=B.hostMatrix;
        this->gpuWorkers=B.gpuWorkers;
    }
}

template <class Toperation>
MatrixMain<Toperation>& MatrixMain<Toperation>::operator*=(MatrixMain<Toperation>& B )
{
    if(this->id==B.id)
    {
        MatrixMain<Toperation>* res;
        {
            MatrixMain<Toperation> aux =B;
            res=&ncclMultEnv->performCalculations(*this,aux,"");
        }
        assignationToActualObject(*res,true,false);
    }else
    {
        MatrixMain<Toperation>& res=ncclMultEnv->performCalculations(*this,B,id);
        assignationToActualObject(res,true,false);
    }
    return *this;
}

template <class Toperation>
MatrixMain<Toperation>& MatrixMain<Toperation>::operator*(MatrixMain<Toperation>& B)
{
    if(B.id==this->id)
    {
        MatrixMain<Toperation>* res;
        {
            MatrixMain<Toperation> aux =B;
            res=&ncclMultEnv->performCalculations(*this,aux,"");
        }
        return *res;
    }
    return ncclMultEnv->performCalculations(*this,B,"");
}

template <class Toperation>
MatrixMain<Toperation>& MatrixMain<Toperation>::operator*=(const Toperation& alpha)
{
    OperationType opType= ncclMultEnv->getOperationType();
    if(isDistributed)
    {   
        int i,j,idPhysicGpu;
        for(i=0;i<gpuWorkers.size();i++)
        {
            idPhysicGpu=gpuWorkers[i]->getGpuRankSystem();
            CUDACHECK(hipSetDevice(idPhysicGpu));
            for(j=0;j<gpuWorkers[i]->getMatricesLocal().size();j++)
            {
                MatrixUtilitiesCuda<Toperation>::scalarCublas(ncclMultEnv->getCublasHandlers()[idPhysicGpu],opType,blockRowSize, blockColumnSize,gpuWorkers[i]->getMatrixLocal(j),alpha,1);
            }
        }
        ncclMultEnv->waitAllCublasStreams();
        setIsMatrixHostHere(false);
    }else
    {
        throw std::invalid_argument("La matriz nos esta distribuida. Realice una multiplicación entre matrices antes.");
    }
    return *this;
}

template <class Toperation>
MatrixMain<Toperation> MatrixMain<Toperation>::operator*(const Toperation& alpha)
{
    MatrixMain<Toperation> aux =*this;
    aux*=alpha;
    return aux;
}

template <class Toperation>
MatrixMain<Toperation>& MatrixMain<Toperation>::operator=(const MatrixMain<Toperation>& B)
{
    assignationToActualObject(B,true,true);
    return *this;
}

template <class Toperation>
MatrixMain<Toperation>& MatrixMain<Toperation>::operator+=(const Toperation& constantAddition)
{
    if(this->getColumnsReal()!=this->getRowsReal())
    {
        throw std::invalid_argument("La operación no se trata de una matriz cuadrada.");
    }
    OperationType opType= ncclMultEnv->getOperationType();
    if(isDistributed)
    {   
        int i,j,idPhysicGpu;
        Toperation* constantAdditionGpu;
        std::vector<Toperation*> constantAdditionGpus;
        for(i=0;i<ncclMultEnv->getGpuSizeOperationSystem();i++)
        {
            idPhysicGpu=gpuWorkers[i]->getGpuRankSystem();
            CUDACHECK(hipSetDevice(idPhysicGpu));
            CUDACHECK(hipMalloc((void**)&constantAdditionGpu,sizeof(Toperation)));
            CUDACHECK(hipMemcpy(constantAdditionGpu,&constantAddition,sizeof(Toperation),hipMemcpyHostToDevice));
            constantAdditionGpus.push_back(constantAdditionGpu);
        }
        for(i=0;i<gpuWorkers.size();i++)
        {
            idPhysicGpu=gpuWorkers[i]->getGpuRankSystem();
            CUDACHECK(hipSetDevice(idPhysicGpu));
            for(j=0;j<gpuWorkers[i]->getMatricesLocal().size();j++)
            {
                //Falta decidir a partir de que indice se hace dentro de la matriz local. 
                //Tambien tendria que averiguar cual es su tamaño real del bloque en vez del usado ya que podria restar o sumar posiciones de 0. Mirar distirbucion o recuperacion
                //este if esta mal. Las posiciones donde empiezan estan mal localmente, bien globalmente
                if(blocksInitialPositionDiagonal[i]!=-1)
                {
                    MatrixUtilitiesCuda<Toperation>::axpyCublas(ncclMultEnv->getCublasHandlers()[idPhysicGpu],opType,blockRowSize, blockColumnSize,constantAdditionGpus[idPhysicGpu],&gpuWorkers[i]->getMatrixLocal(j)[0],1,0,blockRowSize+1);
                }
            }
        }
        ncclMultEnv->waitAllCublasStreams();
        for(i=0;i<ncclMultEnv->getGpuSizeOperationSystem();i++)
        {
            idPhysicGpu=gpuWorkers[i]->getGpuRankSystem();
            CUDACHECK(hipFree(constantAdditionGpus[i]));
        }
        setIsMatrixHostHere(false);
        std::cout<<"K VIENE"<<std::endl;
        MatrixUtilitiesCuda<Toperation>::cudaPrintOneMatrixCall(blockRowSize,blockColumnSize,gpuWorkers[3]->getMatrixLocal(0),opType);
    }else
    {
        throw std::invalid_argument("La matriz nos esta distribuida. Realice una multiplicación entre matrices antes.");
    }
    return *this;
}




template class MatrixMain<double>;
template class MatrixMain<float>;