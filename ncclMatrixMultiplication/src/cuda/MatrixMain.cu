#include "hip/hip_runtime.h"
#include "MatrixMain.cuh"

template <class Toperation>
MatrixMain<Toperation>::MatrixMain(NcclMultiplicationEnvironment<Toperation>* ncclMultEnv,std::string id,int rows,int columns)
{
    this->ncclMultEnv=ncclMultEnv;
    this->id=id;
    this->rowsReal=rows;
    this->columnsReal=columns;
    this->isMatrixHostHere=false;
    this->isDistributed=false;
    this->deleteMatrixHostAtDestroyment=false;
    this->hostMatrix=nullptr;
    if(id=="")
    {
        this->id=this->ncclMultEnv->generateRandomId();
    }
    this->ncclMultEnv->setOrAddMatrixMain(this->id,this);
}

template <class Toperation>
MatrixMain<Toperation>::MatrixMain(NcclMultiplicationEnvironment<Toperation>* ncclMultEnv,std::string id,int rows,int columns,Toperation* matrix):MatrixMain(ncclMultEnv,id,rows,columns)
{
    this->ncclMultEnv=ncclMultEnv;
    this->hostMatrix=matrix;
    this->isMatrixHostHere=true;
}

template <class Toperation>
MatrixMain<Toperation>::~MatrixMain()
{
    if(isMatrixHostHere && deleteMatrixHostAtDestroyment)
    {
        MatrixUtilities<Toperation>::matrixFree(hostMatrix);
    }
    int i;
    for(i=0;i<gpuWorkers.size();i++)
    {
        delete gpuWorkers[i];
    }
    gpuWorkers.clear();
}

template <class Toperation>
std::string  MatrixMain<Toperation>::getId()
{
    return id;
}
template <class Toperation>
int MatrixMain<Toperation>::getRowsReal()
{
    return rowsReal;
}

template <class Toperation>
int MatrixMain<Toperation>::getColumnsReal()
{
    return columnsReal;
}

template <class Toperation>
int MatrixMain<Toperation>::getRowsUsed()
{
    return rowsUsed;
}

template <class Toperation>
int MatrixMain<Toperation>::getColumnsUsed()
{
    return columnsUsed;
}

template <class Toperation>
bool MatrixMain<Toperation>::getIsDistributed()
{
    return isDistributed;
}

template <class Toperation>
int MatrixMain<Toperation>::getBlockSize()
{
    return blockSize;
}

template <class Toperation>
int MatrixMain<Toperation>::getBlockRowSize()
{
    return blockRowSize;
}

template <class Toperation>
int MatrixMain<Toperation>::getBlockColumnSize()
{
    return blockColumnSize;
}

template <class Toperation>
bool MatrixMain<Toperation>::getIsMatrixHostHere()
{
    return isMatrixHostHere;
}

template <class Toperation>
bool MatrixMain<Toperation>::getDeleteMatrixHostAtDestroyment()
{
    return deleteMatrixHostAtDestroyment;
}

template <class Toperation>
Toperation *MatrixMain<Toperation>::getHostMatrix()
{
    if(!isMatrixHostHere)
    {
        recoverMatrixToHost();
    }
    return hostMatrix;
}
template <class Toperation>
std::vector<GpuWorker<Toperation>*> MatrixMain<Toperation>::getGpuWorkers()
{
    return gpuWorkers;
}

template <class Toperation>
void MatrixMain<Toperation>::setId(std::string id)
{
    //Quitar la anterior id del entorno
    this->ncclMultEnv->removeMatrixMain(this->id,false);
    this->id=id;
    //Agregar la nueva id al entorno
    this->ncclMultEnv->setOrAddMatrixMain(id,this);
}

template <class Toperation>
void MatrixMain<Toperation>::setRowsUsed(int rowsUsed)
{
    this->rowsUsed = rowsUsed;
}

template <class Toperation>
void MatrixMain<Toperation>::setColumnsUsed(int columnsUsed)
{
    this->columnsUsed = columnsUsed;
}

template <class Toperation>
void MatrixMain<Toperation>::setIsDistributed(bool isDistributed)
{
    this->isDistributed = isDistributed;
}

template <class Toperation>
void MatrixMain<Toperation>::setIsMatrixHostHere(bool isMatrixHostHere)
{
    this->isMatrixHostHere = isMatrixHostHere;
}

template <class Toperation>
void MatrixMain<Toperation>::setDeleteMatrixHostAtDestroyment(bool deleteMatrixHostAtDestroyment)
{
    this->deleteMatrixHostAtDestroyment=deleteMatrixHostAtDestroyment;
}


template <class Toperation>
void MatrixMain<Toperation>::setMatrixOperationProperties(int meshRowSize, int meshColumnSize, int blockRowSize, int blockColumnSize)
{
    this->meshRowSize=meshRowSize;
    this->meshColumnSize=meshColumnSize;
    this->blockRowSize=blockRowSize;
    this->blockColumnSize=blockColumnSize;
    this->numberOfRowBlocks = ceil(this->rowsUsed / this->blockRowSize);
    this->numberOfColumnBlocks = ceil(this->columnsUsed / this->blockColumnSize);
    this->numberOfTotalBlocks = this->numberOfRowBlocks * this->numberOfColumnBlocks;
    this->blockSize = this->blockRowSize * this->blockColumnSize;
    blocksInitialPosition.resize(numberOfTotalBlocks);
    int i, posColumnBelong, posRowBelong,indexBlock,gpuRealId;
    for (i = 0,indexBlock=0; i < numberOfTotalBlocks; i++)
    {
        posColumnBelong = (i % numberOfColumnBlocks) * rowsReal * blockColumnSize;
        posRowBelong = (i / numberOfColumnBlocks) * blockRowSize;
        blocksInitialPosition[i]=(posColumnBelong + posRowBelong);
        //Debido a ColumnMajorOrder corrijo al indice del bloque que pertenece para una correcta formación de la malla.
        indexBlock=(indexBlock+numberOfColumnBlocks);
        if(indexBlock>=numberOfTotalBlocks){
            indexBlock%=(numberOfTotalBlocks-1);
        }
        //Creacion de los gpuWorkers y su primer bloque
        if(i<ncclMultEnv->getGpuSizeOperationWorld())
        {
            gpuRealId=MatrixUtilitiesCuda<Toperation>::getRealGpuId(i,ncclMultEnv->getGpuSizeSystem());
            GpuWorker<Toperation> *gpuW= new GpuWorker<Toperation>(i,gpuRealId,this);
            gpuWorkers.push_back(gpuW);
            hipStream_t *newStream = new hipStream_t;
            CUDACHECK(hipStreamCreate(newStream));
            gpuWorkers[i]->addStream(newStream);
            Toperation *newMatrix=MatrixUtilitiesCuda<Toperation>::cudaMatrixMemoryAllocation(blockRowSize,blockColumnSize,newStream);
            gpuWorkers[i]->setMatrixLocal(newMatrix);
        }
    }
}

template <class Toperation>
int MatrixMain<Toperation>::calculateRowColor(int gpuRank)
{
    return gpuRank / numberOfColumnBlocks;
}

template <class Toperation>
int MatrixMain<Toperation>::calculateColumnColor(int gpuRank)
{
    return gpuRank % numberOfColumnBlocks;
}

template <class Toperation>
int MatrixMain<Toperation>::calculateBlockDimensionToCopy(int color, int meshDimensionSize, int blockDimenensionSize, int dimensionUsed, int dimensionReal)
{
    return (color != (meshDimensionSize - 1)) ? blockDimenensionSize : (blockDimenensionSize - (dimensionUsed - dimensionReal));
}

template <class Toperation>
void MatrixMain<Toperation>::waitAllStreamsOfAllWorkers()
{
    int i;
    for(i=0;i<gpuWorkers.size();i++)
    {
        CUDACHECK(hipSetDevice(gpuWorkers[i]->getGpuRankSystem()));
        gpuWorkers[i]->waitAllStreams();
    }
}

template <class Toperation>
void MatrixMain<Toperation>::distributeMatrixIntoGpus()
{
    int i,j,k,blockColumnSizeCopy,blockRowSizeCopy;
    for(i=0;i<ncclMultEnv->getGpuSizeOperationWorld()&&i<numberOfTotalBlocks;i++)
    {
        CUDACHECK(hipSetDevice(gpuWorkers[i]->getGpuRankSystem()));
        for(j=i;j<numberOfTotalBlocks;j+=ncclMultEnv->getGpuSizeOperationWorld())
        {
            Toperation *newMatrix;
            hipStream_t *newStream;
            if(j!=i)//El primer bloque ya estaba creado de la llamada a setMatrixOperationProperties
            {
                newStream = new hipStream_t;
                CUDACHECK(hipStreamCreate(newStream));
                gpuWorkers[i]->addStream(newStream);
                newMatrix=MatrixUtilitiesCuda<Toperation>::cudaMatrixMemoryAllocation(blockRowSize,blockColumnSize,newStream);
                gpuWorkers[i]->setMatrixLocal(newMatrix);
            }else 
            {
                newStream=gpuWorkers[i]->getStream(0);
                newMatrix=gpuWorkers[i]->getMatrixLocal(0);
            }
            blockColumnSizeCopy = calculateBlockDimensionToCopy(calculateColumnColor(i), numberOfColumnBlocks, blockColumnSize, columnsUsed, columnsReal);
            blockRowSizeCopy = calculateBlockDimensionToCopy(calculateRowColor(i), numberOfRowBlocks, blockRowSize, rowsUsed, rowsReal);
            for(k=0;k<blockColumnSizeCopy;k++)
            {
                CUDACHECK(hipMemcpyAsync(&newMatrix[k*blockRowSize],&hostMatrix[blocksInitialPosition[j]+k*rowsReal],blockRowSizeCopy*sizeof(Toperation),hipMemcpyHostToDevice,*newStream));
            }
        }
    }
    setIsDistributed(true);
}

template <class Toperation>
void MatrixMain<Toperation>::recoverMatrixToHost()
{
    //OJO QUE SI LA MTRIZ TIENE EN UN WORKER VARIAS MATRICESLOCALES NO VA BIEN. O ESO CREO. TAMPOCO SE SI SE PUEDE DAR EL CASO
    int i,j,k,blockColumnSizeCopy,blockRowSizeCopy;
    hostMatrix=MatrixUtilities<Toperation>::matrixMemoryAllocation(rowsReal,columnsReal);
    for(i=0;i<ncclMultEnv->getGpuSizeOperationWorld()&&i<numberOfTotalBlocks;i++)
    {
        CUDACHECK(hipSetDevice(gpuWorkers[i]->getGpuRankSystem()));
        for(j=0;j<numberOfTotalBlocks;j+=ncclMultEnv->getGpuSizeOperationWorld())
        {
            Toperation *newMatrix;
            hipStream_t *newStream;
            newStream=gpuWorkers[i]->getStream(0);
            newMatrix=gpuWorkers[i]->getMatrixLocal(0);
            blockColumnSizeCopy = calculateBlockDimensionToCopy(calculateColumnColor(i), numberOfColumnBlocks, blockColumnSize, columnsUsed, columnsReal);
            blockRowSizeCopy = calculateBlockDimensionToCopy(calculateRowColor(i), numberOfRowBlocks, blockRowSize, rowsUsed, rowsReal);
            for(k=0;k<blockColumnSizeCopy;k++)
            {
                CUDACHECK(hipMemcpyAsync(&hostMatrix[blocksInitialPosition[i]+k*rowsReal],&newMatrix[k*blockRowSize],blockRowSizeCopy*sizeof(Toperation),hipMemcpyDeviceToHost,*newStream));
            }
        }
    }
    waitAllStreamsOfAllWorkers();
    setIsMatrixHostHere(true);
}

template <class Toperation>
MatrixMain<Toperation> MatrixMain<Toperation>::operator*=(MatrixMain<Toperation> B )
{
    /////////////////NO FUNCIONA////////////////////////
    MatrixMain<Toperation> aux=(*this)*B;
    return aux;
}

template <class Toperation>
MatrixMain<Toperation> MatrixMain<Toperation>::operator*(MatrixMain<Toperation> B)
{
    return *(ncclMultEnv->performCalculations(id,B.getId(),""));
}



template class MatrixMain<double>;
template class MatrixMain<float>;