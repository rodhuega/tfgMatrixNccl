#include "hip/hip_runtime.h"
#include "MatrixMain.cuh"

template <class Toperation>
MatrixMain<Toperation>::MatrixMain(NcclMultiplicationEnvironment<Toperation>* ncclMultEnv,std::string id,int rows,int columns)
{
    this->ncclMultEnv=ncclMultEnv;
    this->id=id;
    this->rowsReal=rows;
    this->columnsReal=columns;
    this->isDistributed=false;
    this->ncclMultEnv->setOrAddMatrixMain(id,this);
}

template <class Toperation>
MatrixMain<Toperation>::MatrixMain(NcclMultiplicationEnvironment<Toperation>* ncclMultEnv,std::string id,int rows,int columns,Toperation* matrix):MatrixMain(ncclMultEnv,id,rows,columns)
{
    this->ncclMultEnv=ncclMultEnv;
    this->hostMatrix=matrix;
    this->isMatrixHostHere=true;
}

template <class Toperation>
int MatrixMain<Toperation>::getRowsReal()
{
    return rowsReal;
}

template <class Toperation>
int MatrixMain<Toperation>::getColumnsReal()
{
    return columnsReal;
}

template <class Toperation>
int MatrixMain<Toperation>::getRowsUsed()
{
    return rowsUsed;
}

template <class Toperation>
int MatrixMain<Toperation>::getColumnsUsed()
{
    return columnsUsed;
}

template <class Toperation>
bool MatrixMain<Toperation>::getIsDistributed()
{
    return isDistributed;
}

template <class Toperation>
int MatrixMain<Toperation>::getBlockSize()
{
    return blockSize;
}

template <class Toperation>
int MatrixMain<Toperation>::getBlockRowSize()
{
    return blockRowSize;
}

template <class Toperation>
int MatrixMain<Toperation>::getBlockColumnSize()
{
    return blockColumnSize;
}

template <class Toperation>
bool MatrixMain<Toperation>::getIsMatrixHostHere()
{
    return isMatrixHostHere;
}

template <class Toperation>
Toperation *MatrixMain<Toperation>::getHostMatrix()
{
    return hostMatrix;
}
template <class Toperation>
std::vector<GpuWorker<Toperation>*> MatrixMain<Toperation>::getGpuWorkers()
{
    return gpuWorkers;
}

template <class Toperation>
void MatrixMain<Toperation>::setRowsUsed(int rowsUsed)
{
    this->rowsUsed = rowsUsed;
}

template <class Toperation>
void MatrixMain<Toperation>::setColumnsUsed(int columnsUsed)
{
    this->columnsUsed = columnsUsed;
}

template <class Toperation>
void MatrixMain<Toperation>::setIsDistributed(bool isDistributed)
{
    this->isDistributed = isDistributed;
}

template <class Toperation>
void MatrixMain<Toperation>::setIsMatrixHostHere(bool isMatrixHostHere)
{
    this->isMatrixHostHere = isMatrixHostHere;
}

template <class Toperation>
void MatrixMain<Toperation>::setBlockAndMeshSize(int meshRowSize, int meshColumnSize, int blockRowSize, int blockColumnSize)
{
    this->meshRowSize=meshRowSize;
    this->meshColumnSize=meshColumnSize;
    this->blockRowSize=blockRowSize;
    this->blockColumnSize=blockColumnSize;
    this->numberOfRowBlocks = ceil(this->rowsUsed / this->blockRowSize);
    this->numberOfColumnBlocks = ceil(this->columnsUsed / this->blockColumnSize);
    this->numberOfTotalBlocks = this->numberOfRowBlocks * this->numberOfColumnBlocks;
    this->blockSize = this->blockRowSize * this->blockColumnSize;

    int i, posColumnBelong, posRowBelong;
    for (i = 0; i < numberOfTotalBlocks; i++)
    {
        posRowBelong = (i / meshColumnSize) * columnsReal * blockRowSize;
        posColumnBelong = (i % meshColumnSize) * blockColumnSize;
        blocksInitialPosition.push_back(posColumnBelong + posRowBelong);
    }
}

template <class Toperation>
int MatrixMain<Toperation>::calculateRowColor(int gpuRank)
{
    return gpuRank / numberOfColumnBlocks;
}

template <class Toperation>
int MatrixMain<Toperation>::calculateColumnColor(int gpuRank)
{
    return gpuRank % numberOfColumnBlocks;
}

template <class Toperation>
int MatrixMain<Toperation>::calculateBlockDimensionToCopy(int color, int meshDimensionSize, int blockDimenensionSize, int dimensionUsed, int dimensionReal)
{
    return (color != (meshDimensionSize - 1)) ? blockDimenensionSize : (blockDimenensionSize - (dimensionUsed - dimensionReal));
}

template <class Toperation>
void MatrixMain<Toperation>::waitAllStreamsOfAllWorkers()
{
    int i;
    for(i=0;i<gpuWorkers.size();i++)
    {
        CUDACHECK(hipSetDevice(gpuWorkers[i]->getGpuRankSystem()));
        gpuWorkers[i]->waitAllStreams();
    }
}

template <class Toperation>
void MatrixMain<Toperation>::distributeMatrixIntoGpus()
{
    int i,j,k,blockColumnSizeCopy,blockRowSizeCopy;
    for(i=0;i<ncclMultEnv->getGpuSizeOperationWorld();i++)
    {
        int gpuRealId=MatrixUtilitiesCuda<Toperation>::getRealGpuId(i,ncclMultEnv->getGpuSizeSystem());
        GpuWorker<Toperation> *gpuW= new GpuWorker<Toperation>(i,gpuRealId,this);
        gpuWorkers.push_back(gpuW);
        CUDACHECK(hipSetDevice(gpuWorkers[i]->getGpuRankSystem()));
        for(j=0;j<numberOfTotalBlocks;j+=ncclMultEnv->getGpuSizeOperationWorld())
        {
            hipStream_t *newStream = new hipStream_t;
            CUDACHECK(hipStreamCreate(newStream));
            gpuWorkers[i]->addStream(newStream);
            Toperation *newMatrix=MatrixUtilitiesCuda<Toperation>::cudaMatrixMemoryAllocation(blockRowSize,blockColumnSize,newStream);
            
            blockColumnSizeCopy = calculateBlockDimensionToCopy(calculateColumnColor(i), numberOfColumnBlocks, blockColumnSize, columnsUsed, columnsReal);
            blockRowSizeCopy = calculateBlockDimensionToCopy(calculateRowColor(i), numberOfRowBlocks, blockRowSize, rowsUsed, rowsReal);
            for(k=0;k<blockColumnSizeCopy;k++)
            {
                //W.I.P Indice blocksInitialPosition puede que este mal
                CUDACHECK(hipMemcpyAsync(&newMatrix[k*blockRowSize],&hostMatrix[blocksInitialPosition[i]+k*columnsReal],blockRowSizeCopy*sizeof(Toperation),hipMemcpyHostToDevice,*newStream));
            }
            gpuWorkers[i]->setMatrixLocal(newMatrix);
        }
    }
}


template class MatrixMain<double>;