#include "hip/hip_runtime.h"
#include <stdio.h>
#include <fstream>
#include <random>
#include <unistd.h>
#include <vector>
#include <iomanip>
#include <algorithm>
#include "hip/hip_runtime.h"
#include <hipblas.h>
#include "nccl.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#ifdef CUBLAS_API_H_
// cuBLAS API errors
static const char *_cudaGetErrorEnum(hipblasStatus_t error)
{
    switch (error)
    {
        case HIPBLAS_STATUS_SUCCESS:
            return "HIPBLAS_STATUS_SUCCESS";

        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "HIPBLAS_STATUS_NOT_INITIALIZED";

        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "HIPBLAS_STATUS_ALLOC_FAILED";

        case HIPBLAS_STATUS_INVALID_VALUE:
            return "HIPBLAS_STATUS_INVALID_VALUE";

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "HIPBLAS_STATUS_ARCH_MISMATCH";

        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "HIPBLAS_STATUS_MAPPING_ERROR";

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "HIPBLAS_STATUS_EXECUTION_FAILED";

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "HIPBLAS_STATUS_INTERNAL_ERROR";
    }

    return "<unknown>";
}
#endif


#define IDX2C(i,j,ld) (((j)*(ld))+(i))

#define CUDACHECK(cmd)                                         \
	do                                                         \
	{                                                          \
		hipError_t e = cmd;                                   \
		if (e != hipSuccess)                                  \
		{                                                      \
			printf("Failed: Cuda error %s:%d '%s'\n",          \
				   __FILE__, __LINE__, hipGetErrorString(e)); \
			exit(EXIT_FAILURE);                                \
		}                                                      \
	} while (0)

#define CUBLASCHECK(cmd)                                         \
do                                                         \
{                                                          \
	hipblasStatus_t s = cmd;                                   \
	if (s != HIPBLAS_STATUS_SUCCESS)                                  \
	{                                                      \
		printf("Failed: Cublas error %s:%d '%s'\n",          \
				__FILE__, __LINE__, _cudaGetErrorEnum(s)); \
		exit(EXIT_FAILURE);                                \
	}                                                      \
} while (0)

#define NCCLCHECK(cmd)                                         \
	do                                                         \
	{                                                          \
		ncclResult_t r = cmd;                                  \
		if (r != ncclSuccess)                                  \
		{                                                      \
			printf("Failed, NCCL error %s:%d '%s'\n",          \
				   __FILE__, __LINE__, ncclGetErrorString(r)); \
			exit(EXIT_FAILURE);                                \
		}                                                      \
	} while (0)

#define cudaCalloc(A, B, C) \
    do { \
        hipError_t __cudaCalloc_err = hipMalloc(A, B*C); \
        if (__cudaCalloc_err == hipSuccess) CUDACHECK(hipMemset(*A, 0, B*C)); \
    } while (0)

using namespace std;


struct OperationProperties
{
    /**
     * @brief Tamaño de las filas de la malla
     * 
     */
    int meshRowSize;
    /**
     * @brief Tamaño de las columnas de la malla
     * 
     */
    int meshColumnSize;
    /**
     * @brief Filas operacionales que tendra la matriz A
     * 
     */
    int rowsA;
    /**
     * @brief Columnas de la Matriz A o Filas de la Matriz B operacionales
     * 
     */
    int columnsAorRowsB;
    /**
     * @brief Columnas operacionales que tendra la matriz B
     * 
     */
    int columnsB;
    /**
     * @brief Numeros de 0s que tendra la matriz operacional al extenderse
     * 
     */
    int numberOf0;
    /**
     * @brief Numero de procesadores que realizaran la operacion de multiplicacion
     * 
     */
    int cpuSize;
    /**
     * @brief Numero de filas que tendra la matriz A de forma local
     * 
     */
    int blockRowSizeA;
    /**
     * @brief Numero de columnas que tendra la matriz A de forma local
     * 
     */
    int blockColumnSizeA;
    /**
     * @brief Numero de filas que tendra la matriz B de forma local
     * 
     */
    int blockRowSizeB;
    /**
     * @brief Numero de columnas que tendra la matriz B de forma local
     * 
     */
    int blockColumnSizeB;
    /**
     * @brief Incida si las propiedades antes indicadas son aptas para el calculo de la multiplicaicon de la matriz
     * 
     */
    bool candidate;
};

struct GpuProperties
{
	int nDevicesGlobal;
	int nDevicesOperation;
	int rankGlobal;
	int rankOperation;
	int rankRow;
	int rankCol;
	int *devicesGlobal;
	int *devicesOperation;
	int *devicesRow;
	int *devicesCol;
	ncclComm_t *commGlobal;
	ncclComm_t *commOperation;
	ncclComm_t *commRow;
	ncclComm_t *commCol;
	hipStream_t *streams;
	hipblasHandle_t handle;
	double *matrixDeviceA;
	double *matrixDeviceB;
	double *matrixDeviceC;
	int rowSize;
	int colSize;

	GpuProperties(int devicesTotal, int rankGlobal)
	{
		this->nDevicesGlobal = devicesTotal;
		this->rankGlobal = rankGlobal;
	}
};

int matrixCalculateIndex(int columnSize, int rowIndex, int columnIndex)
{
	return columnSize * rowIndex + columnIndex;
}

void printMatrix(int rows, int columns, double *M)
{
    int i, j, matrixIndex;
    for (i = 0; i < rows; ++i)
    {
        for (j = 0; j < columns; ++j)
        {
            matrixIndex = IDX2C(i, j,rows);
            cout << M[matrixIndex] << "\t";
        }
        cout << endl;
    }
}

double* matrixMemoryAllocation(int rows, int columns)
{
	double *matrix = (double *)calloc(rows * columns, sizeof(double));
	return matrix;
}


void matrixFree(double *matrix)
{
	free(matrix);
}




double *ReadOrGenerateRandomMatrix(bool isRandom, const char *fileName, int &rows, int &columns, int boundLower, int boundUpper)
{
	int i, j, matrixIndex;
	std::ifstream file;
	if (!isRandom)
	{
		file.open(fileName);
		file >> rows >> columns;
	}
	//Configuracion del generador de numeros por si se genera una matriz random
	random_device rd;
	mt19937 eng(rd());
	uniform_real_distribution<> distr(boundLower, boundUpper);
	double *matrix = matrixMemoryAllocation(rows, columns);
	//Bucle de generacion o lectura de la matrizs
	for (i = 0; i < rows; i++)
	{
		for (j = 0; j < columns; j++)
		{
			matrixIndex = IDX2C(i,j,rows);
			if (isRandom)
			{
				matrix[matrixIndex] = distr(eng);
			}
			else
			{
				file >> matrix[matrixIndex];
			}
		}
	}
	if (!isRandom)
	{
		file.close();
	}
	return matrix;
}

OperationProperties calculateNonEqualMesh(int rowsA, int columnsAorRowsB, int columnsB, int nCpusMesh1, int nCpusMesh2, bool isMeshRow)
{
    OperationProperties res;
    if (isMeshRow)
    {
        res.meshRowSize = nCpusMesh1;
        res.meshColumnSize = nCpusMesh2;
    }
    else
    {
        res.meshColumnSize = nCpusMesh1;
        res.meshRowSize = nCpusMesh2;
    }
    res.cpuSize = res.meshRowSize * res.meshColumnSize;
    res.rowsA = ceil(rowsA / (float)res.meshRowSize) * res.meshRowSize;
    res.columnsAorRowsB = ceil(columnsAorRowsB / (float)res.meshColumnSize) * res.meshColumnSize;
    res.columnsB = ceil(columnsB / (float)res.meshColumnSize) * res.meshColumnSize;
    int numberOf0atA = (res.rowsA * res.columnsAorRowsB) - (rowsA * columnsAorRowsB);
    int numberOf0atB = (res.columnsB * res.columnsAorRowsB) - (columnsAorRowsB * columnsB);
    res.numberOf0 = numberOf0atA + numberOf0atB;
    //PUEDE QUE AQUI NECESITE UN IF DEPENDIENDO DE CUAL SEA EL GRID DOMINANTE; DE MOMENTO EL GRID DOMINANTE AHORA ES A SIEMPRE
    res.blockColumnSizeA = res.columnsAorRowsB / res.meshColumnSize;
    res.blockRowSizeB = res.blockColumnSizeA;
    res.blockRowSizeA = res.rowsA / res.meshRowSize;
    res.blockColumnSizeB = res.columnsB / res.meshColumnSize;
    res.candidate = res.meshColumnSize > 1 && res.meshRowSize > 1;
    return res;
}

OperationProperties getMeshAndMatrixSize(int rowsA, int columnsA, int rowsB, int columnsB, int cpuSize)
{
    OperationProperties res;

    //Se calculan todas las posibilidadades y se selecciona la que mas cpus use y menos 0 contenga de esas opciones, Solo se añaden elementos validos(ninguno con meshDimension 1 o 0)
    int i, j, numberOfZerosA, numberOfZerosB;
    vector<OperationProperties> allOp;
    vector<OperationProperties> sameCpuSizeOp;
    for (i = 2; i < cpuSize - 1; i++)
    {
        for (j = i; j * i <= cpuSize; j++)
        {
            OperationProperties opRow = calculateNonEqualMesh(rowsA, rowsB, columnsB, i, j, true);
            OperationProperties opColumn = calculateNonEqualMesh(rowsA, rowsB, columnsB, i, j, false);
            if (opRow.candidate)
            {
                allOp.push_back(opRow);
            }
            if (opColumn.candidate)
            {
                allOp.push_back(opColumn);
            }
        }
    }
    sort(begin(allOp), end(allOp), [](OperationProperties op1, OperationProperties op2) {
        if (op1.cpuSize != op2.cpuSize)
        {
            return op1.cpuSize > op2.cpuSize;
        }
        return op1.numberOf0 < op2.numberOf0;
    });
    res = allOp[0];

    return res;
}


__global__ void
cudaPrintMatrix(int rows,int columns,double* matrix)
{
	for(int i =0;i<rows;i++)
	{
		for(int j=0;j<columns;j++)
		{
			printf("%.2lf\t",matrix[IDX2C(i,j,rows)]);
		}
		printf("\n");
	}
}


int main(int argc, char *argv[])
{
	cout << fixed;
    cout << setprecision(2);
	//Leer parametros del lanzamiento del programa
	bool printMatrixBool = false;
	vector<string> optionsCmd;
	int rowsA,columnsA,i;
	double *matrixA;
	for (i = 0; i < argc; i++)
	{
		optionsCmd.push_back(string(argv[i]));
	}
	if (std::find(optionsCmd.begin(), optionsCmd.end(), "-h") != optionsCmd.end() || optionsCmd.size() == 1)
	{
		cout << "Uso:\tLas opciones -f y -r no se pueden usar a la vez" << endl;
		cout << "\t-h\tMuestra la ayuda" << endl;
		cout << "\t-p\t(Opcional) Muestra la matriz por pantalla" << endl;
		cout << "\t-f\tLas matrices son leidas de ficheros de texto: -f f1.txt" << endl;
		cout << "\t-r\tLas matrices son generadas de forma aleatoria(m n indican el tamaño de las matrices. bl bu indican de que numero a que numero se genera la matrix .Todos son numeros enteros) -r m n" << endl;
	}
	if (std::find(optionsCmd.begin(), optionsCmd.end(), "-p") != optionsCmd.end())
	{
		printMatrixBool = true;
	}
	auto fOptionChecker = std::find(optionsCmd.begin(), optionsCmd.end(), "-f");
	auto rOptionChecker = std::find(optionsCmd.begin(), optionsCmd.end(), "-r");
	if (fOptionChecker != optionsCmd.end() && rOptionChecker != optionsCmd.end())
	{
		cout << "Los parametros -f y -r no se pueden usar a la vez" << endl;
		return -1;
	}
	if (fOptionChecker != optionsCmd.end())
	{
		int fPosition = std::distance(optionsCmd.begin(), fOptionChecker);
		matrixA = ReadOrGenerateRandomMatrix(false, optionsCmd[fPosition + 1].c_str(), rowsA, columnsA, -1, -1);
	}

	if (rOptionChecker != optionsCmd.end())
	{
		int rPosition = std::distance(optionsCmd.begin(), rOptionChecker);
		rowsA=atoi(optionsCmd[rPosition + 1].c_str());
		columnsA=atoi(optionsCmd[rPosition + 2].c_str());
		matrixA = ReadOrGenerateRandomMatrix(true, "", rowsA, columnsA, atoi(optionsCmd[rPosition + 3].c_str()), atoi(optionsCmd[rPosition + 4].c_str()));
	}
	printMatrix(rowsA,columnsA,matrixA);
	int nDevicesGlobal;
	CUDACHECK(hipGetDeviceCount(&nDevicesGlobal));
	OperationProperties op = getMeshAndMatrixSize(rowsA, columnsA, rowsA, columnsA, nDevicesGlobal);
	//Configuracion del comunicador que tiene a todos los dispositivos
	
	
	GpuProperties *gpusInfo = (GpuProperties *)malloc(sizeof(GpuProperties) * nDevicesGlobal);

	int devicesGlobal[nDevicesGlobal];
	for (int i = 0; i < nDevicesGlobal; i++)
	{

        int posRowBelong = (i / op.meshColumnSize) * op.blockRowSizeA;
		int posColumnBelong = (i % op.meshColumnSize) * op.blockColumnSizeA;
		cout<<"blockRowSizeA:"<<op.blockRowSizeA<<", blockColumnSizeA: "<<op.blockColumnSizeA<<", Primer elemento: "<< matrixA[IDX2C(posRowBelong,posColumnBelong,rowsA)]<<endl;
		CUDACHECK(hipSetDevice(i));
		gpusInfo[i] = GpuProperties(nDevicesGlobal, i);
		gpusInfo[i].streams = (hipStream_t *)malloc(sizeof(hipStream_t*)*2);
		CUDACHECK(hipStreamCreate(&gpusInfo[i].streams[0]));
		CUDACHECK(hipStreamCreate(&gpusInfo[i].streams[1]));
		devicesGlobal[i] = i;
		CUDACHECK(hipMalloc ((void**)&gpusInfo[i].matrixDeviceA, op.blockRowSizeA*op.blockColumnSizeA*sizeof(double)));
		CUDACHECK(hipMalloc ((void**)&gpusInfo[i].matrixDeviceB, op.blockRowSizeA*op.blockColumnSizeA*sizeof(double)));
		CUDACHECK(hipMalloc ((void**)&gpusInfo[i].matrixDeviceC, op.blockRowSizeA*op.blockColumnSizeA*sizeof(double)));
		CUBLASCHECK(hipblasSetMatrix(op.blockRowSizeA, op.blockColumnSizeA, sizeof(double), &matrixA[IDX2C(posRowBelong,posColumnBelong,rowsA)], rowsA, gpusInfo[i].matrixDeviceA, op.blockRowSizeA));
		CUBLASCHECK(hipblasCreate(&gpusInfo[i].handle));
		cudaPrintMatrix<<<1,1,1>>>(op.blockRowSizeA,op.blockColumnSizeA,gpusInfo[i].matrixDeviceA);
		hipDeviceSynchronize();
		cout<<endl;
	}

	ncclComm_t commGlobal[nDevicesGlobal];

	//initializing NCCL
	NCCLCHECK(ncclCommInitAll(commGlobal, nDevicesGlobal, devicesGlobal));

	//Liberar memoria
	for (int i = 0; i < nDevicesGlobal; ++i)
		ncclCommDestroy(commGlobal[i]);
	free(gpusInfo);
	std::cout << "Fin del programa" << std::endl;
	return 0;
}