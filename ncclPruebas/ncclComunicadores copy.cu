#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "nccl.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#define CUDACHECK(cmd)                                         \
	do                                                         \
	{                                                          \
		hipError_t e = cmd;                                   \
		if (e != hipSuccess)                                  \
		{                                                      \
			printf("Failed: Cuda error %s:%d '%s'\n",          \
				   __FILE__, __LINE__, hipGetErrorString(e)); \
			exit(EXIT_FAILURE);                                \
		}                                                      \
	} while (0)

#define NCCLCHECK(cmd)                                         \
	do                                                         \
	{                                                          \
		ncclResult_t r = cmd;                                  \
		if (r != ncclSuccess)                                  \
		{                                                      \
			printf("Failed, NCCL error %s:%d '%s'\n",          \
				   __FILE__, __LINE__, ncclGetErrorString(r)); \
			exit(EXIT_FAILURE);                                \
		}                                                      \
	} while (0)

struct GpuProperties
{
	int nDevicesGlobal;
	int nDevicesOperation;
	int rankGlobal;
	int rankOperation;
	int rankRow;
	int rankCol;
	int *devicesGlobal;
	int *devicesOperation;
	int *devicesRow;
	int *devicesCol;
	ncclComm_t *commGlobal;
	ncclComm_t *commOperation;
	ncclComm_t *commRow;
	ncclComm_t *commCol;
	hipStream_t *stream;

	GpuProperties(int devicesTotal,int rankGlobal)
	{
		this->nDevicesGlobal=devicesTotal;
		this->rankGlobal=rankGlobal;
	}

};

__global__ void
cudaHelloWorld()
{
	printf("Hola\n");
}

int main(int argc, char *argv[])
{
	//Saber cuantas graficas tengo en el sistema
	int nDevicesGlobal;
	CUDACHECK(hipGetDeviceCount(&nDevicesGlobal));
	//Creacion del comunicador global
	int devicesGlobal[nDevicesGlobal];
	std::cout<<"Seguimos vivos1"<<std::endl;
	GpuProperties *gpusInfo =(GpuProperties*)malloc(sizeof(gpusInfo)*nDevicesGlobal);
	std::cout<<"Seguimos vivos2"<<std::endl;

	for (int i = 0; i < nDevicesGlobal; i++)
	{
		gpusInfo[i]=GpuProperties(nDevicesGlobal,i);
		devicesGlobal[i] = i;
	}
	std::cout<<"Seguimos vivos3"<<std::endl;

	ncclComm_t commGlobal[nDevicesGlobal];
	
	
	hipStream_t *s = (hipStream_t *)malloc(sizeof(hipStream_t) * nDevicesGlobal);

	//initializing NCCL
	NCCLCHECK(ncclCommInitAll(commGlobal, nDevicesGlobal, devicesGlobal));

	//finalizing NCCL
	// for (int i = 0; i < nDevicesGlobal; ++i)
	// 	ncclCommDestroy(commGlobal[i]);
	// free(gpusInfo);
	std::cout << "Fin del programa" << std::endl;
	return 0;
}